#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_fp16.h>
#include <distributed.h>

// kernal declaration
__global__
void divergence(int n, 
              int steps,
              double delta_t,
              bool *still_together,
              bool *not_diverged,
              int *times,
              double m_1, double m_2, double m_3,
              double critical_distance,
              double *p1_x, double *p1_y, double *p1_z, 
              double *p2_x, double *p2_y, double *p2_z, 
              double *p3_x, double *p3_y, double *p3_z, 
              double *p1_prime_x, double *p1_prime_y, double *p1_prime_z, 
              double *p2_prime_x, double *p2_prime_y, double *p2_prime_z, 
              double *p3_prime_x, double *p3_prime_y, double *p3_prime_z, 
              double *dv_1_x, double *dv_1_y, double *dv_1_z,
              double *dv_2_x, double *dv_2_y, double *dv_2_z,
              double *dv_3_x, double *dv_3_y, double *dv_3_z,
              double *dv_1pr_x, double *dv_1pr_y, double *dv_1pr_z,
              double *dv_2pr_x, double *dv_2pr_y, double *dv_2pr_z,
              double *dv_3pr_x, double *dv_3pr_y, double *dv_3pr_z,
              double *v1_x, double *v1_y, double *v1_z,
              double *v2_x, double *v2_y, double *v2_z,
              double *v3_x, double *v3_y, double *v3_z,
              double *v1_prime_x, double *v1_prime_y, double *v1_prime_z,
              double *v2_prime_x, double *v2_prime_y, double *v2_prime_z,
              double *v3_prime_x, double *v3_prime_y, double *v3_prime_z,
              double *nv1_x, double *nv1_y, double *nv1_z,
              double *nv2_x, double *nv2_y, double *nv2_z,
              double *nv3_x, double *nv3_y, double *nv3_z,
              double *nv1_prime_x, double *nv1_prime_y, double *nv1_prime_z,
              double *nv2_prime_x, double *nv2_prime_y, double *nv2_prime_z,
              double *nv3_prime_x, double *nv3_prime_y, double *nv3_prime_z
              )
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  for (int j=0; j < steps; j++) {
    if (i < n and still_together[i]){
      // compute accelerations
      dv_1_x[i] = -9.8 * m_2 * (p1_x[i] - p2_x[i]) / (sqrt((p1_x[i] - p2_x[i])*(p1_x[i] - p2_x[i]) + (p1_y[i] - p2_y[i])*(p1_y[i] - p2_y[i]) + (p1_z[i] - p2_z[i])*(p1_z[i] - p2_z[i]))*sqrt((p1_x[i] - p2_x[i])*(p1_x[i] - p2_x[i]) + (p1_y[i] - p2_y[i])*(p1_y[i] - p2_y[i]) + (p1_z[i] - p2_z[i])*(p1_z[i] - p2_z[i]))*sqrt((p1_x[i] - p2_x[i])*(p1_x[i] - p2_x[i]) + (p1_y[i] - p2_y[i])*(p1_y[i] - p2_y[i]) + (p1_z[i] - p2_z[i])*(p1_z[i] - p2_z[i]))) -9.8 * m_3 * (p1_x[i] - p3_x[i]) / (sqrt((p1_x[i] - p3_x[i])*(p1_x[i] - p3_x[i]) + (p1_y[i] - p3_y[i])*(p1_y[i] - p3_y[i]) + (p1_z[i] - p3_z[i])*(p1_z[i] - p3_z[i]))*sqrt((p1_x[i] - p3_x[i])*(p1_x[i] - p3_x[i]) + (p1_y[i] - p3_y[i])*(p1_y[i] - p3_y[i]) + (p1_z[i] - p3_z[i])*(p1_z[i] - p3_z[i]))*sqrt((p1_x[i] - p3_x[i])*(p1_x[i] - p3_x[i]) + (p1_y[i] - p3_y[i])*(p1_y[i] - p3_y[i]) + (p1_z[i] - p3_z[i])*(p1_z[i] - p3_z[i])));
      dv_1_y[i] = -9.8 * m_2 * (p1_y[i] - p2_y[i]) / (sqrt((p1_x[i] - p2_x[i])*(p1_x[i] - p2_x[i]) + (p1_y[i] - p2_y[i])*(p1_y[i] - p2_y[i]) + (p1_z[i] - p2_z[i])*(p1_z[i] - p2_z[i]))*sqrt((p1_x[i] - p2_x[i])*(p1_x[i] - p2_x[i]) + (p1_y[i] - p2_y[i])*(p1_y[i] - p2_y[i]) + (p1_z[i] - p2_z[i])*(p1_z[i] - p2_z[i]))*sqrt((p1_x[i] - p2_x[i])*(p1_x[i] - p2_x[i]) + (p1_y[i] - p2_y[i])*(p1_y[i] - p2_y[i]) + (p1_z[i] - p2_z[i])*(p1_z[i] - p2_z[i]))) -9.8 * m_3 * (p1_y[i] - p3_y[i]) / (sqrt((p1_x[i] - p3_x[i])*(p1_x[i] - p3_x[i]) + (p1_y[i] - p3_y[i])*(p1_y[i] - p3_y[i]) + (p1_z[i] - p3_z[i])*(p1_z[i] - p3_z[i]))*sqrt((p1_x[i] - p3_x[i])*(p1_x[i] - p3_x[i]) + (p1_y[i] - p3_y[i])*(p1_y[i] - p3_y[i]) + (p1_z[i] - p3_z[i])*(p1_z[i] - p3_z[i]))*sqrt((p1_x[i] - p3_x[i])*(p1_x[i] - p3_x[i]) + (p1_y[i] - p3_y[i])*(p1_y[i] - p3_y[i]) + (p1_z[i] - p3_z[i])*(p1_z[i] - p3_z[i])));
      dv_1_z[i] = -9.8 * m_2 * (p1_z[i] - p2_z[i]) / (sqrt((p1_x[i] - p2_x[i])*(p1_x[i] - p2_x[i]) + (p1_y[i] - p2_y[i])*(p1_y[i] - p2_y[i]) + (p1_z[i] - p2_z[i])*(p1_z[i] - p2_z[i]))*sqrt((p1_x[i] - p2_x[i])*(p1_x[i] - p2_x[i]) + (p1_y[i] - p2_y[i])*(p1_y[i] - p2_y[i]) + (p1_z[i] - p2_z[i])*(p1_z[i] - p2_z[i]))*sqrt((p1_x[i] - p2_x[i])*(p1_x[i] - p2_x[i]) + (p1_y[i] - p2_y[i])*(p1_y[i] - p2_y[i]) + (p1_z[i] - p2_z[i])*(p1_z[i] - p2_z[i]))) -9.8 * m_3 * (p1_z[i] - p3_z[i]) / (sqrt((p1_x[i] - p3_x[i])*(p1_x[i] - p3_x[i]) + (p1_y[i] - p3_y[i])*(p1_y[i] - p3_y[i]) + (p1_z[i] - p3_z[i])*(p1_z[i] - p3_z[i]))*sqrt((p1_x[i] - p3_x[i])*(p1_x[i] - p3_x[i]) + (p1_y[i] - p3_y[i])*(p1_y[i] - p3_y[i]) + (p1_z[i] - p3_z[i])*(p1_z[i] - p3_z[i]))*sqrt((p1_x[i] - p3_x[i])*(p1_x[i] - p3_x[i]) + (p1_y[i] - p3_y[i])*(p1_y[i] - p3_y[i]) + (p1_z[i] - p3_z[i])*(p1_z[i] - p3_z[i])));
      
      dv_2_x[i] = -9.8 * m_3 * (p2_x[i] - p3_x[i]) / (sqrt((p2_x[i] - p3_x[i])*(p2_x[i] - p3_x[i]) + (p2_y[i] - p3_y[i])*(p2_y[i] - p3_y[i]) + (p2_z[i] - p3_z[i])*(p2_z[i] - p3_z[i]))*sqrt((p2_x[i] - p3_x[i])*(p2_x[i] - p3_x[i]) + (p2_y[i] - p3_y[i])*(p2_y[i] - p3_y[i]) + (p2_z[i] - p3_z[i])*(p2_z[i] - p3_z[i]))*sqrt((p2_x[i] - p3_x[i])*(p2_x[i] - p3_x[i]) + (p2_y[i] - p3_y[i])*(p2_y[i] - p3_y[i]) + (p2_z[i] - p3_z[i])*(p2_z[i] - p3_z[i]))) -9.8 * m_1 * (p2_x[i] - p1_x[i]) / (sqrt((p2_x[i] - p1_x[i])*(p2_x[i] - p1_x[i]) + (p2_y[i] - p1_y[i])*(p2_y[i] - p1_y[i]) + (p2_z[i] - p1_z[i])*(p2_z[i] - p1_z[i]))*sqrt((p2_x[i] - p1_x[i])*(p2_x[i] - p1_x[i]) + (p2_y[i] - p1_y[i])*(p2_y[i] - p1_y[i]) + (p2_z[i] - p1_z[i])*(p2_z[i] - p1_z[i]))*sqrt((p2_x[i] - p1_x[i])*(p2_x[i] - p1_x[i]) + (p2_y[i] - p1_y[i])*(p2_y[i] - p1_y[i]) + (p2_z[i] - p1_z[i])*(p2_z[i] - p1_z[i])));
      dv_2_y[i] = -9.8 * m_3 * (p2_y[i] - p3_y[i]) / (sqrt((p2_x[i] - p3_x[i])*(p2_x[i] - p3_x[i]) + (p2_y[i] - p3_y[i])*(p2_y[i] - p3_y[i]) + (p2_z[i] - p3_z[i])*(p2_z[i] - p3_z[i]))*sqrt((p2_x[i] - p3_x[i])*(p2_x[i] - p3_x[i]) + (p2_y[i] - p3_y[i])*(p2_y[i] - p3_y[i]) + (p2_z[i] - p3_z[i])*(p2_z[i] - p3_z[i]))*sqrt((p2_x[i] - p3_x[i])*(p2_x[i] - p3_x[i]) + (p2_y[i] - p3_y[i])*(p2_y[i] - p3_y[i]) + (p2_z[i] - p3_z[i])*(p2_z[i] - p3_z[i]))) -9.8 * m_1 * (p2_y[i] - p1_y[i]) / (sqrt((p2_x[i] - p1_x[i])*(p2_x[i] - p1_x[i]) + (p2_y[i] - p1_y[i])*(p2_y[i] - p1_y[i]) + (p2_z[i] - p1_z[i])*(p2_z[i] - p1_z[i]))*sqrt((p2_x[i] - p1_x[i])*(p2_x[i] - p1_x[i]) + (p2_y[i] - p1_y[i])*(p2_y[i] - p1_y[i]) + (p2_z[i] - p1_z[i])*(p2_z[i] - p1_z[i]))*sqrt((p2_x[i] - p1_x[i])*(p2_x[i] - p1_x[i]) + (p2_y[i] - p1_y[i])*(p2_y[i] - p1_y[i]) + (p2_z[i] - p1_z[i])*(p2_z[i] - p1_z[i])));
      dv_2_z[i] = -9.8 * m_3 * (p2_z[i] - p3_z[i]) / (sqrt((p2_x[i] - p3_x[i])*(p2_x[i] - p3_x[i]) + (p2_y[i] - p3_y[i])*(p2_y[i] - p3_y[i]) + (p2_z[i] - p3_z[i])*(p2_z[i] - p3_z[i]))*sqrt((p2_x[i] - p3_x[i])*(p2_x[i] - p3_x[i]) + (p2_y[i] - p3_y[i])*(p2_y[i] - p3_y[i]) + (p2_z[i] - p3_z[i])*(p2_z[i] - p3_z[i]))*sqrt((p2_x[i] - p3_x[i])*(p2_x[i] - p3_x[i]) + (p2_y[i] - p3_y[i])*(p2_y[i] - p3_y[i]) + (p2_z[i] - p3_z[i])*(p2_z[i] - p3_z[i]))) -9.8 * m_1 * (p2_z[i] - p1_z[i]) / (sqrt((p2_x[i] - p1_x[i])*(p2_x[i] - p1_x[i]) + (p2_y[i] - p1_y[i])*(p2_y[i] - p1_y[i]) + (p2_z[i] - p1_z[i])*(p2_z[i] - p1_z[i]))*sqrt((p2_x[i] - p1_x[i])*(p2_x[i] - p1_x[i]) + (p2_y[i] - p1_y[i])*(p2_y[i] - p1_y[i]) + (p2_z[i] - p1_z[i])*(p2_z[i] - p1_z[i]))*sqrt((p2_x[i] - p1_x[i])*(p2_x[i] - p1_x[i]) + (p2_y[i] - p1_y[i])*(p2_y[i] - p1_y[i]) + (p2_z[i] - p1_z[i])*(p2_z[i] - p1_z[i])));

      dv_3_x[i] = -9.8 * m_1 * (p3_x[i] - p1_x[i]) / (sqrt((p3_x[i] - p1_x[i])*(p3_x[i] - p1_x[i]) + (p3_y[i] - p1_y[i])*(p3_y[i] - p1_y[i]) + (p3_z[i] - p1_z[i])*(p3_z[i] - p1_z[i]))*sqrt((p3_x[i] - p1_x[i])*(p3_x[i] - p1_x[i]) + (p3_y[i] - p1_y[i])*(p3_y[i] - p1_y[i]) + (p3_z[i] - p1_z[i])*(p3_z[i] - p1_z[i]))*sqrt((p3_x[i] - p1_x[i])*(p3_x[i] - p1_x[i]) + (p3_y[i] - p1_y[i])*(p3_y[i] - p1_y[i]) + (p3_z[i] - p1_z[i])*(p3_z[i] - p1_z[i]))) -9.8 * m_2 * (p3_x[i] - p2_x[i]) / (sqrt((p3_x[i] - p2_x[i])*(p3_x[i] - p2_x[i]) + (p3_y[i] - p2_y[i])*(p3_y[i] - p2_y[i]) + (p3_z[i] - p2_z[i])*(p3_z[i] - p2_z[i]))*sqrt((p3_x[i] - p2_x[i])*(p3_x[i] - p2_x[i]) + (p3_y[i] - p2_y[i])*(p3_y[i] - p2_y[i]) + (p3_z[i] - p2_z[i])*(p3_z[i] - p2_z[i]))*sqrt((p3_x[i] - p2_x[i])*(p3_x[i] - p2_x[i]) + (p3_y[i] - p2_y[i])*(p3_y[i] - p2_y[i]) + (p3_z[i] - p2_z[i])*(p3_z[i] - p2_z[i])));
      dv_3_y[i] = -9.8 * m_1 * (p3_y[i] - p1_y[i]) / (sqrt((p3_x[i] - p1_x[i])*(p3_x[i] - p1_x[i]) + (p3_y[i] - p1_y[i])*(p3_y[i] - p1_y[i]) + (p3_z[i] - p1_z[i])*(p3_z[i] - p1_z[i]))*sqrt((p3_x[i] - p1_x[i])*(p3_x[i] - p1_x[i]) + (p3_y[i] - p1_y[i])*(p3_y[i] - p1_y[i]) + (p3_z[i] - p1_z[i])*(p3_z[i] - p1_z[i]))*sqrt((p3_x[i] - p1_x[i])*(p3_x[i] - p1_x[i]) + (p3_y[i] - p1_y[i])*(p3_y[i] - p1_y[i]) + (p3_z[i] - p1_z[i])*(p3_z[i] - p1_z[i]))) -9.8 * m_2 * (p3_y[i] - p2_y[i]) / (sqrt((p3_x[i] - p2_x[i])*(p3_x[i] - p2_x[i]) + (p3_y[i] - p2_y[i])*(p3_y[i] - p2_y[i]) + (p3_z[i] - p2_z[i])*(p3_z[i] - p2_z[i]))*sqrt((p3_x[i] - p2_x[i])*(p3_x[i] - p2_x[i]) + (p3_y[i] - p2_y[i])*(p3_y[i] - p2_y[i]) + (p3_z[i] - p2_z[i])*(p3_z[i] - p2_z[i]))*sqrt((p3_x[i] - p2_x[i])*(p3_x[i] - p2_x[i]) + (p3_y[i] - p2_y[i])*(p3_y[i] - p2_y[i]) + (p3_z[i] - p2_z[i])*(p3_z[i] - p2_z[i])));
      dv_3_z[i] = -9.8 * m_1 * (p3_z[i] - p1_z[i]) / (sqrt((p3_x[i] - p1_x[i])*(p3_x[i] - p1_x[i]) + (p3_y[i] - p1_y[i])*(p3_y[i] - p1_y[i]) + (p3_z[i] - p1_z[i])*(p3_z[i] - p1_z[i]))*sqrt((p3_x[i] - p1_x[i])*(p3_x[i] - p1_x[i]) + (p3_y[i] - p1_y[i])*(p3_y[i] - p1_y[i]) + (p3_z[i] - p1_z[i])*(p3_z[i] - p1_z[i]))*sqrt((p3_x[i] - p1_x[i])*(p3_x[i] - p1_x[i]) + (p3_y[i] - p1_y[i])*(p3_y[i] - p1_y[i]) + (p3_z[i] - p1_z[i])*(p3_z[i] - p1_z[i]))) -9.8 * m_2 * (p3_z[i] - p2_z[i]) / (sqrt((p3_x[i] - p2_x[i])*(p3_x[i] - p2_x[i]) + (p3_y[i] - p2_y[i])*(p3_y[i] - p2_y[i]) + (p3_z[i] - p2_z[i])*(p3_z[i] - p2_z[i]))*sqrt((p3_x[i] - p2_x[i])*(p3_x[i] - p2_x[i]) + (p3_y[i] - p2_y[i])*(p3_y[i] - p2_y[i]) + (p3_z[i] - p2_z[i])*(p3_z[i] - p2_z[i]))*sqrt((p3_x[i] - p2_x[i])*(p3_x[i] - p2_x[i]) + (p3_y[i] - p2_y[i])*(p3_y[i] - p2_y[i]) + (p3_z[i] - p2_z[i])*(p3_z[i] - p2_z[i])));

      dv_1pr_x[i] = -9.8 * m_2 * (p1_prime_x[i] - p2_prime_x[i]) / (sqrt((p1_prime_x[i] - p2_prime_x[i])*(p1_prime_x[i] - p2_prime_x[i]) + (p1_prime_y[i] - p2_prime_y[i])*(p1_prime_y[i] - p2_prime_y[i]) + (p1_prime_z[i] - p2_prime_z[i])*(p1_prime_z[i] - p2_prime_z[i]))*sqrt((p1_prime_x[i] - p2_prime_x[i])*(p1_prime_x[i] - p2_prime_x[i]) + (p1_prime_y[i] - p2_prime_y[i])*(p1_prime_y[i] - p2_prime_y[i]) + (p1_prime_z[i] - p2_prime_z[i])*(p1_prime_z[i] - p2_prime_z[i]))*sqrt((p1_prime_x[i] - p2_prime_x[i])*(p1_prime_x[i] - p2_prime_x[i]) + (p1_prime_y[i] - p2_prime_y[i])*(p1_prime_y[i] - p2_prime_y[i]) + (p1_prime_z[i] - p2_prime_z[i])*(p1_prime_z[i] - p2_prime_z[i]))) -9.8 * m_3 * (p1_prime_x[i] - p3_prime_x[i]) / (sqrt((p1_prime_x[i] - p3_prime_x[i])*(p1_prime_x[i] - p3_prime_x[i]) + (p1_prime_y[i] - p3_prime_y[i])*(p1_prime_y[i] - p3_prime_y[i]) + (p1_prime_z[i] - p3_prime_z[i])*(p1_prime_z[i] - p3_prime_z[i]))*sqrt((p1_prime_x[i] - p3_prime_x[i])*(p1_prime_x[i] - p3_prime_x[i]) + (p1_prime_y[i] - p3_prime_y[i])*(p1_prime_y[i] - p3_prime_y[i]) + (p1_prime_z[i] - p3_prime_z[i])*(p1_prime_z[i] - p3_prime_z[i]))*sqrt((p1_prime_x[i] - p3_prime_x[i])*(p1_prime_x[i] - p3_prime_x[i]) + (p1_prime_y[i] - p3_prime_y[i])*(p1_prime_y[i] - p3_prime_y[i]) + (p1_prime_z[i] - p3_prime_z[i])*(p1_prime_z[i] - p3_prime_z[i])));
      dv_1pr_y[i] = -9.8 * m_2 * (p1_prime_y[i] - p2_prime_y[i]) / (sqrt((p1_prime_x[i] - p2_prime_x[i])*(p1_prime_x[i] - p2_prime_x[i]) + (p1_prime_y[i] - p2_prime_y[i])*(p1_prime_y[i] - p2_prime_y[i]) + (p1_prime_z[i] - p2_prime_z[i])*(p1_prime_z[i] - p2_prime_z[i]))*sqrt((p1_prime_x[i] - p2_prime_x[i])*(p1_prime_x[i] - p2_prime_x[i]) + (p1_prime_y[i] - p2_prime_y[i])*(p1_prime_y[i] - p2_prime_y[i]) + (p1_prime_z[i] - p2_prime_z[i])*(p1_prime_z[i] - p2_prime_z[i]))*sqrt((p1_prime_x[i] - p2_prime_x[i])*(p1_prime_x[i] - p2_prime_x[i]) + (p1_prime_y[i] - p2_prime_y[i])*(p1_prime_y[i] - p2_prime_y[i]) + (p1_prime_z[i] - p2_prime_z[i])*(p1_prime_z[i] - p2_prime_z[i]))) -9.8 * m_3 * (p1_prime_y[i] - p3_prime_y[i]) / (sqrt((p1_prime_x[i] - p3_prime_x[i])*(p1_prime_x[i] - p3_prime_x[i]) + (p1_prime_y[i] - p3_prime_y[i])*(p1_prime_y[i] - p3_prime_y[i]) + (p1_prime_z[i] - p3_prime_z[i])*(p1_prime_z[i] - p3_prime_z[i]))*sqrt((p1_prime_x[i] - p3_prime_x[i])*(p1_prime_x[i] - p3_prime_x[i]) + (p1_prime_y[i] - p3_prime_y[i])*(p1_prime_y[i] - p3_prime_y[i]) + (p1_prime_z[i] - p3_prime_z[i])*(p1_prime_z[i] - p3_prime_z[i]))*sqrt((p1_prime_x[i] - p3_prime_x[i])*(p1_prime_x[i] - p3_prime_x[i]) + (p1_prime_y[i] - p3_prime_y[i])*(p1_prime_y[i] - p3_prime_y[i]) + (p1_prime_z[i] - p3_prime_z[i])*(p1_prime_z[i] - p3_prime_z[i])));
      dv_1pr_z[i] = -9.8 * m_2 * (p1_prime_z[i] - p2_prime_z[i]) / (sqrt((p1_prime_x[i] - p2_prime_x[i])*(p1_prime_x[i] - p2_prime_x[i]) + (p1_prime_y[i] - p2_prime_y[i])*(p1_prime_y[i] - p2_prime_y[i]) + (p1_prime_z[i] - p2_prime_z[i])*(p1_prime_z[i] - p2_prime_z[i]))*sqrt((p1_prime_x[i] - p2_prime_x[i])*(p1_prime_x[i] - p2_prime_x[i]) + (p1_prime_y[i] - p2_prime_y[i])*(p1_prime_y[i] - p2_prime_y[i]) + (p1_prime_z[i] - p2_prime_z[i])*(p1_prime_z[i] - p2_prime_z[i]))*sqrt((p1_prime_x[i] - p2_prime_x[i])*(p1_prime_x[i] - p2_prime_x[i]) + (p1_prime_y[i] - p2_prime_y[i])*(p1_prime_y[i] - p2_prime_y[i]) + (p1_prime_z[i] - p2_prime_z[i])*(p1_prime_z[i] - p2_prime_z[i]))) -9.8 * m_3 * (p1_prime_z[i] - p3_prime_z[i]) / (sqrt((p1_prime_x[i] - p3_prime_x[i])*(p1_prime_x[i] - p3_prime_x[i]) + (p1_prime_y[i] - p3_prime_y[i])*(p1_prime_y[i] - p3_prime_y[i]) + (p1_prime_z[i] - p3_prime_z[i])*(p1_prime_z[i] - p3_prime_z[i]))*sqrt((p1_prime_x[i] - p3_prime_x[i])*(p1_prime_x[i] - p3_prime_x[i]) + (p1_prime_y[i] - p3_prime_y[i])*(p1_prime_y[i] - p3_prime_y[i]) + (p1_prime_z[i] - p3_prime_z[i])*(p1_prime_z[i] - p3_prime_z[i]))*sqrt((p1_prime_x[i] - p3_prime_x[i])*(p1_prime_x[i] - p3_prime_x[i]) + (p1_prime_y[i] - p3_prime_y[i])*(p1_prime_y[i] - p3_prime_y[i]) + (p1_prime_z[i] - p3_prime_z[i])*(p1_prime_z[i] - p3_prime_z[i])));
      
      dv_2pr_x[i] = -9.8 * m_3 * (p2_prime_x[i] - p3_prime_x[i]) / (sqrt((p2_prime_x[i] - p3_prime_x[i])*(p2_prime_x[i] - p3_prime_x[i]) + (p2_prime_y[i] - p3_prime_y[i])*(p2_prime_y[i] - p3_prime_y[i]) + (p2_prime_z[i] - p3_prime_z[i])*(p2_prime_z[i] - p3_prime_z[i]))*sqrt((p2_prime_x[i] - p3_prime_x[i])*(p2_prime_x[i] - p3_prime_x[i]) + (p2_prime_y[i] - p3_prime_y[i])*(p2_prime_y[i] - p3_prime_y[i]) + (p2_prime_z[i] - p3_prime_z[i])*(p2_prime_z[i] - p3_prime_z[i]))*sqrt((p2_prime_x[i] - p3_prime_x[i])*(p2_prime_x[i] - p3_prime_x[i]) + (p2_prime_y[i] - p3_prime_y[i])*(p2_prime_y[i] - p3_prime_y[i]) + (p2_prime_z[i] - p3_prime_z[i])*(p2_prime_z[i] - p3_prime_z[i]))) -9.8 * m_1 * (p2_prime_x[i] - p1_prime_x[i]) / (sqrt((p2_prime_x[i] - p1_prime_x[i])*(p2_prime_x[i] - p1_prime_x[i]) + (p2_prime_y[i] - p1_prime_y[i])*(p2_prime_y[i] - p1_prime_y[i]) + (p2_prime_z[i] - p1_prime_z[i])*(p2_prime_z[i] - p1_prime_z[i]))*sqrt((p2_prime_x[i] - p1_prime_x[i])*(p2_prime_x[i] - p1_prime_x[i]) + (p2_prime_y[i] - p1_prime_y[i])*(p2_prime_y[i] - p1_prime_y[i]) + (p2_prime_z[i] - p1_prime_z[i])*(p2_prime_z[i] - p1_prime_z[i]))*sqrt((p2_prime_x[i] - p1_prime_x[i])*(p2_prime_x[i] - p1_prime_x[i]) + (p2_prime_y[i] - p1_prime_y[i])*(p2_prime_y[i] - p1_prime_y[i]) + (p2_prime_z[i] - p1_prime_z[i])*(p2_prime_z[i] - p1_prime_z[i])));
      dv_2pr_y[i] = -9.8 * m_3 * (p2_prime_y[i] - p3_prime_y[i]) / (sqrt((p2_prime_x[i] - p3_prime_x[i])*(p2_prime_x[i] - p3_prime_x[i]) + (p2_prime_y[i] - p3_prime_y[i])*(p2_prime_y[i] - p3_prime_y[i]) + (p2_prime_z[i] - p3_prime_z[i])*(p2_prime_z[i] - p3_prime_z[i]))*sqrt((p2_prime_x[i] - p3_prime_x[i])*(p2_prime_x[i] - p3_prime_x[i]) + (p2_prime_y[i] - p3_prime_y[i])*(p2_prime_y[i] - p3_prime_y[i]) + (p2_prime_z[i] - p3_prime_z[i])*(p2_prime_z[i] - p3_prime_z[i]))*sqrt((p2_prime_x[i] - p3_prime_x[i])*(p2_prime_x[i] - p3_prime_x[i]) + (p2_prime_y[i] - p3_prime_y[i])*(p2_prime_y[i] - p3_prime_y[i]) + (p2_prime_z[i] - p3_prime_z[i])*(p2_prime_z[i] - p3_prime_z[i]))) -9.8 * m_1 * (p2_prime_y[i] - p1_prime_y[i]) / (sqrt((p2_prime_x[i] - p1_prime_x[i])*(p2_prime_x[i] - p1_prime_x[i]) + (p2_prime_y[i] - p1_prime_y[i])*(p2_prime_y[i] - p1_prime_y[i]) + (p2_prime_z[i] - p1_prime_z[i])*(p2_prime_z[i] - p1_prime_z[i]))*sqrt((p2_prime_x[i] - p1_prime_x[i])*(p2_prime_x[i] - p1_prime_x[i]) + (p2_prime_y[i] - p1_prime_y[i])*(p2_prime_y[i] - p1_prime_y[i]) + (p2_prime_z[i] - p1_prime_z[i])*(p2_prime_z[i] - p1_prime_z[i]))*sqrt((p2_prime_x[i] - p1_prime_x[i])*(p2_prime_x[i] - p1_prime_x[i]) + (p2_prime_y[i] - p1_prime_y[i])*(p2_prime_y[i] - p1_prime_y[i]) + (p2_prime_z[i] - p1_prime_z[i])*(p2_prime_z[i] - p1_prime_z[i])));
      dv_2pr_z[i] = -9.8 * m_3 * (p2_prime_z[i] - p3_prime_z[i]) / (sqrt((p2_prime_x[i] - p3_prime_x[i])*(p2_prime_x[i] - p3_prime_x[i]) + (p2_prime_y[i] - p3_prime_y[i])*(p2_prime_y[i] - p3_prime_y[i]) + (p2_prime_z[i] - p3_prime_z[i])*(p2_prime_z[i] - p3_prime_z[i]))*sqrt((p2_prime_x[i] - p3_prime_x[i])*(p2_prime_x[i] - p3_prime_x[i]) + (p2_prime_y[i] - p3_prime_y[i])*(p2_prime_y[i] - p3_prime_y[i]) + (p2_prime_z[i] - p3_prime_z[i])*(p2_prime_z[i] - p3_prime_z[i]))*sqrt((p2_prime_x[i] - p3_prime_x[i])*(p2_prime_x[i] - p3_prime_x[i]) + (p2_prime_y[i] - p3_prime_y[i])*(p2_prime_y[i] - p3_prime_y[i]) + (p2_prime_z[i] - p3_prime_z[i])*(p2_prime_z[i] - p3_prime_z[i]))) -9.8 * m_1 * (p2_prime_z[i] - p1_prime_z[i]) / (sqrt((p2_prime_x[i] - p1_prime_x[i])*(p2_prime_x[i] - p1_prime_x[i]) + (p2_prime_y[i] - p1_prime_y[i])*(p2_prime_y[i] - p1_prime_y[i]) + (p2_prime_z[i] - p1_prime_z[i])*(p2_prime_z[i] - p1_prime_z[i]))*sqrt((p2_prime_x[i] - p1_prime_x[i])*(p2_prime_x[i] - p1_prime_x[i]) + (p2_prime_y[i] - p1_prime_y[i])*(p2_prime_y[i] - p1_prime_y[i]) + (p2_prime_z[i] - p1_prime_z[i])*(p2_prime_z[i] - p1_prime_z[i]))*sqrt((p2_prime_x[i] - p1_prime_x[i])*(p2_prime_x[i] - p1_prime_x[i]) + (p2_prime_y[i] - p1_prime_y[i])*(p2_prime_y[i] - p1_prime_y[i]) + (p2_prime_z[i] - p1_prime_z[i])*(p2_prime_z[i] - p1_prime_z[i])));

      dv_3pr_x[i] = -9.8 * m_1 * (p3_prime_x[i] - p1_prime_x[i]) / (sqrt((p3_prime_x[i] - p1_prime_x[i])*(p3_prime_x[i] - p1_prime_x[i]) + (p3_prime_y[i] - p1_prime_y[i])*(p3_prime_y[i] - p1_prime_y[i]) + (p3_prime_z[i] - p1_prime_z[i])*(p3_prime_z[i] - p1_prime_z[i]))*sqrt((p3_prime_x[i] - p1_prime_x[i])*(p3_prime_x[i] - p1_prime_x[i]) + (p3_prime_y[i] - p1_prime_y[i])*(p3_prime_y[i] - p1_prime_y[i]) + (p3_prime_z[i] - p1_prime_z[i])*(p3_prime_z[i] - p1_prime_z[i]))*sqrt((p3_prime_x[i] - p1_prime_x[i])*(p3_prime_x[i] - p1_prime_x[i]) + (p3_prime_y[i] - p1_prime_y[i])*(p3_prime_y[i] - p1_prime_y[i]) + (p3_prime_z[i] - p1_prime_z[i])*(p3_prime_z[i] - p1_prime_z[i]))) -9.8 * m_2 * (p3_prime_x[i] - p2_prime_x[i]) / (sqrt((p3_prime_x[i] - p2_prime_x[i])*(p3_prime_x[i] - p2_prime_x[i]) + (p3_prime_y[i] - p2_prime_y[i])*(p3_prime_y[i] - p2_prime_y[i]) + (p3_prime_z[i] - p2_prime_z[i])*(p3_prime_z[i] - p2_prime_z[i]))*sqrt((p3_prime_x[i] - p2_prime_x[i])*(p3_prime_x[i] - p2_prime_x[i]) + (p3_prime_y[i] - p2_prime_y[i])*(p3_prime_y[i] - p2_prime_y[i]) + (p3_prime_z[i] - p2_prime_z[i])*(p3_prime_z[i] - p2_prime_z[i]))*sqrt((p3_prime_x[i] - p2_prime_x[i])*(p3_prime_x[i] - p2_prime_x[i]) + (p3_prime_y[i] - p2_prime_y[i])*(p3_prime_y[i] - p2_prime_y[i]) + (p3_prime_z[i] - p2_prime_z[i])*(p3_prime_z[i] - p2_prime_z[i])));
      dv_3pr_y[i] = -9.8 * m_1 * (p3_prime_y[i] - p1_prime_y[i]) / (sqrt((p3_prime_x[i] - p1_prime_x[i])*(p3_prime_x[i] - p1_prime_x[i]) + (p3_prime_y[i] - p1_prime_y[i])*(p3_prime_y[i] - p1_prime_y[i]) + (p3_prime_z[i] - p1_prime_z[i])*(p3_prime_z[i] - p1_prime_z[i]))*sqrt((p3_prime_x[i] - p1_prime_x[i])*(p3_prime_x[i] - p1_prime_x[i]) + (p3_prime_y[i] - p1_prime_y[i])*(p3_prime_y[i] - p1_prime_y[i]) + (p3_prime_z[i] - p1_prime_z[i])*(p3_prime_z[i] - p1_prime_z[i]))*sqrt((p3_prime_x[i] - p1_prime_x[i])*(p3_prime_x[i] - p1_prime_x[i]) + (p3_prime_y[i] - p1_prime_y[i])*(p3_prime_y[i] - p1_prime_y[i]) + (p3_prime_z[i] - p1_prime_z[i])*(p3_prime_z[i] - p1_prime_z[i]))) -9.8 * m_2 * (p3_prime_y[i] - p2_prime_y[i]) / (sqrt((p3_prime_x[i] - p2_prime_x[i])*(p3_prime_x[i] - p2_prime_x[i]) + (p3_prime_y[i] - p2_prime_y[i])*(p3_prime_y[i] - p2_prime_y[i]) + (p3_prime_z[i] - p2_prime_z[i])*(p3_prime_z[i] - p2_prime_z[i]))*sqrt((p3_prime_x[i] - p2_prime_x[i])*(p3_prime_x[i] - p2_prime_x[i]) + (p3_prime_y[i] - p2_prime_y[i])*(p3_prime_y[i] - p2_prime_y[i]) + (p3_prime_z[i] - p2_prime_z[i])*(p3_prime_z[i] - p2_prime_z[i]))*sqrt((p3_prime_x[i] - p2_prime_x[i])*(p3_prime_x[i] - p2_prime_x[i]) + (p3_prime_y[i] - p2_prime_y[i])*(p3_prime_y[i] - p2_prime_y[i]) + (p3_prime_z[i] - p2_prime_z[i])*(p3_prime_z[i] - p2_prime_z[i])));
      dv_3pr_z[i] = -9.8 * m_1 * (p3_prime_z[i] - p1_prime_z[i]) / (sqrt((p3_prime_x[i] - p1_prime_x[i])*(p3_prime_x[i] - p1_prime_x[i]) + (p3_prime_y[i] - p1_prime_y[i])*(p3_prime_y[i] - p1_prime_y[i]) + (p3_prime_z[i] - p1_prime_z[i])*(p3_prime_z[i] - p1_prime_z[i]))*sqrt((p3_prime_x[i] - p1_prime_x[i])*(p3_prime_x[i] - p1_prime_x[i]) + (p3_prime_y[i] - p1_prime_y[i])*(p3_prime_y[i] - p1_prime_y[i]) + (p3_prime_z[i] - p1_prime_z[i])*(p3_prime_z[i] - p1_prime_z[i]))*sqrt((p3_prime_x[i] - p1_prime_x[i])*(p3_prime_x[i] - p1_prime_x[i]) + (p3_prime_y[i] - p1_prime_y[i])*(p3_prime_y[i] - p1_prime_y[i]) + (p3_prime_z[i] - p1_prime_z[i])*(p3_prime_z[i] - p1_prime_z[i]))) -9.8 * m_2 * (p3_prime_z[i] - p2_prime_z[i]) / (sqrt((p3_prime_x[i] - p2_prime_x[i])*(p3_prime_x[i] - p2_prime_x[i]) + (p3_prime_y[i] - p2_prime_y[i])*(p3_prime_y[i] - p2_prime_y[i]) + (p3_prime_z[i] - p2_prime_z[i])*(p3_prime_z[i] - p2_prime_z[i]))*sqrt((p3_prime_x[i] - p2_prime_x[i])*(p3_prime_x[i] - p2_prime_x[i]) + (p3_prime_y[i] - p2_prime_y[i])*(p3_prime_y[i] - p2_prime_y[i]) + (p3_prime_z[i] - p2_prime_z[i])*(p3_prime_z[i] - p2_prime_z[i]))*sqrt((p3_prime_x[i] - p2_prime_x[i])*(p3_prime_x[i] - p2_prime_x[i]) + (p3_prime_y[i] - p2_prime_y[i])*(p3_prime_y[i] - p2_prime_y[i]) + (p3_prime_z[i] - p2_prime_z[i])*(p3_prime_z[i] - p2_prime_z[i])));


      // find which trajectories have diverged and increment *times
      not_diverged[i] = (p1_x[i]-p1_prime_x[i])*(p1_x[i]-p1_prime_x[i]) + (p1_y[i]-p1_prime_y[i])*(p1_y[i]-p1_prime_y[i]) + (p1_z[i]-p1_prime_z[i])*(p1_z[i]-p1_prime_z[i]) <= critical_distance*critical_distance;
      still_together[i] = not_diverged[i] & still_together[i]; // bitwise and 
      if (still_together[i] == 1){
        times[i]++;
      };

      // compute new velocities
      nv1_x[i] = v1_x[i] + delta_t * dv_1_x[i];
      nv1_y[i] = v1_y[i] + delta_t * dv_1_y[i];
      nv1_z[i] = v1_z[i] + delta_t * dv_1_z[i];

      nv2_x[i] = v2_x[i] + delta_t * dv_2_x[i];
      nv2_y[i] = v2_y[i] + delta_t * dv_2_y[i];
      nv2_z[i] = v2_z[i] + delta_t * dv_2_z[i];

      nv3_x[i] = v3_x[i] + delta_t * dv_3_x[i];
      nv3_y[i] = v3_y[i] + delta_t * dv_3_y[i];
      nv3_z[i] = v3_z[i] + delta_t * dv_3_z[i];

      nv1_prime_x[i] = v1_prime_x[i] + delta_t * dv_1pr_x[i];
      nv1_prime_y[i] = v1_prime_y[i] + delta_t * dv_1pr_y[i];
      nv1_prime_z[i] = v1_prime_z[i] + delta_t * dv_1pr_z[i];

      nv2_prime_x[i] = v2_prime_x[i] + delta_t * dv_2pr_x[i];
      nv2_prime_y[i] = v2_prime_y[i] + delta_t * dv_2pr_y[i];
      nv2_prime_z[i] = v2_prime_z[i] + delta_t * dv_2pr_z[i];

      nv3_prime_x[i] = v3_prime_x[i] + delta_t * dv_3pr_x[i];
      nv3_prime_y[i] = v3_prime_y[i] + delta_t * dv_3pr_y[i];
      nv3_prime_z[i] = v3_prime_z[i] + delta_t * dv_3pr_z[i];

      // compute positions with current velocities
      p1_x[i] = p1_x[i] + delta_t * v1_x[i];
      p1_y[i] = p1_y[i] + delta_t * v1_y[i];
      p1_z[i] = p1_z[i] + delta_t * v1_z[i];

      p2_x[i] = p2_x[i] + delta_t * v2_x[i];
      p2_y[i] = p2_y[i] + delta_t * v2_y[i];
      p2_z[i] = p2_z[i] + delta_t * v2_z[i];

      p3_x[i] = p3_x[i] + delta_t * v3_x[i];
      p3_y[i] = p3_y[i] + delta_t * v3_y[i];
      p3_z[i] = p3_z[i] + delta_t * v3_z[i];

      p1_prime_x[i] = p1_prime_x[i] + delta_t * v1_prime_x[i];
      p1_prime_y[i] = p1_prime_y[i] + delta_t * v1_prime_y[i];
      p1_prime_z[i] = p1_prime_z[i] + delta_t * v1_prime_z[i];

      p2_prime_x[i] = p2_prime_x[i] + delta_t * v2_prime_x[i];
      p2_prime_y[i] = p2_prime_y[i] + delta_t * v2_prime_y[i];
      p2_prime_z[i] = p2_prime_z[i] + delta_t * v2_prime_z[i];

      p3_prime_x[i] = p3_prime_x[i] + delta_t * v3_prime_x[i];
      p3_prime_y[i] = p3_prime_y[i] + delta_t * v3_prime_y[i];
      p3_prime_z[i] = p3_prime_z[i] + delta_t * v3_prime_z[i];

      // assign new velocities to current velocities
      v1_x[i] = nv1_x[i];
      v1_y[i] = nv1_y[i];
      v1_z[i] = nv1_z[i];

      v2_x[i] = nv2_x[i];
      v2_y[i] = nv2_y[i];
      v2_z[i] = nv2_z[i];

      v3_x[i] = nv3_x[i];
      v3_y[i] = nv3_y[i];
      v3_z[i] = nv3_z[i];

      v1_prime_x[i] = nv1_prime_x[i];
      v1_prime_y[i] = nv1_prime_y[i];
      v1_prime_z[i] = nv1_prime_z[i];

      v2_prime_x[i] = nv2_prime_x[i];
      v2_prime_y[i] = nv2_prime_y[i];
      v2_prime_z[i] = nv2_prime_z[i];

      v3_prime_x[i] = nv3_prime_x[i];
      v3_prime_y[i] = nv3_prime_y[i];
      v3_prime_z[i] = nv3_prime_z[i];
      }
    }
  }