
#include <stdio.h>
#include <iostream>
#include <chrono>
#include <fstream>
#include <iterator>
#include <fstream>
#include <omp.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


__global__
void saxpy(int n, float a, float *x)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  for (int j=0; j < 10000000; j++){
    if (i < n) {
      x[i] += 1;
    }
  }
}

int main(void) {
  std::chrono::time_point<std::chrono::system_clock> start, end;
  start = std::chrono::system_clock::now();
  float *x, *d_x;

  int N = 10000000;
  hipHostAlloc((void**)&x, N*sizeof(float), hipHostMallocWriteCombined | hipHostMallocMapped);
  // y = (float*)malloc(N*sizeof(float));
  // x = (float*)malloc(N*sizeof(float));

  int n_gpus=4;
  for (int i = 0; i < N/n_gpus; i++) {
      x[i] = 1.0f;
  }

  hipStream_t streams[n_gpus];
  #pragma omp parallel num_threads(n_gpus)
  {
    int d=omp_get_thread_num();
    std::cout << "Thread" << d << "activated\n";
    std::cout << d << " Device initialized \n";
    int start_idx = (N/n_gpus)*omp_get_thread_num();
    int end_idx = start_idx + (N/n_gpus);
    std::cout << "Start index: " << start_idx << "\n";
    std::cout << "End index: " << end_idx << "\n";
    hipSetDevice(omp_get_thread_num());
    hipStreamCreate(&streams[d]);

    hipMalloc(&d_x, (N/n_gpus)*sizeof(float));
    hipMemcpyAsync(d_x, x+start_idx, (N/n_gpus)*sizeof(float), hipMemcpyHostToDevice, streams[d]);

    saxpy<<<(N+127)/128, 128, 0, streams[d]>>>(N/n_gpus, 2.0f, d_x);

    hipMemcpyAsync(x+start_idx, d_x, (N/n_gpus)*sizeof(float), hipMemcpyDeviceToHost, streams[d]);
    hipDeviceSynchronize();
  }

//  cudaDeviceSynchronize();
  hipError_t err = hipGetLastError();  // add
  if (err != hipSuccess) std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl; // add
  std::cout << x[0] << "\n";

  // measure elapsed kernal runtime
  end = std::chrono::system_clock::now();
  std::chrono::duration<double> elapsed_seconds = end - start;
  std::time_t end_time = std::chrono::system_clock::to_time_t(end);
  std::cout << "Elapsed Time: " << elapsed_seconds.count() << "s\n";

  return;
}

// extern "C" {
// float* save_arr()
//   {
//     int N = 1<<20;
//     float *x, *y, *d_x, *d_y;
//     x = (float*)malloc(N*sizeof(float));
//     y = (float*)malloc(N*sizeof(float));

//     cudaMalloc(&d_x, N*sizeof(float)); 
//     cudaMalloc(&d_y, N*sizeof(float));

//     for (int i = 0; i < N; i++) {
//       x[i] = 1.0f;
//       y[i] = 2.0f;
//     }

//     cudaMemcpy(d_x, x, N*sizeof(float), cudaMemcpyHostToDevice);
//     cudaMemcpy(d_y, y, N*sizeof(float), cudaMemcpyHostToDevice);

//     std::chrono::time_point<std::chrono::system_clock> start, end;
//     start = std::chrono::system_clock::now();

//     // Perform SAXPY on 1M elements
//     saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);

//     cudaDeviceSynchronize();

//     // measure elapsed kernal runtime
//     end = std::chrono::system_clock::now();
//     std::chrono::duration<double> elapsed_seconds = end - start;
//     std::time_t end_time = std::chrono::system_clock::to_time_t(end);
//     std::cout << "Elapsed Time: " << elapsed_seconds.count() << "s\n";

//     cudaMemcpy(y, d_y, N*sizeof(float), cudaMemcpyDeviceToHost);

//     cudaFree(d_x);
//     cudaFree(d_y);
//     free(x);
//     return y;
//   }
// }
