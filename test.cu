#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <chrono>


__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  for (int j=0; j < 1000; j++){
    if (i < n) {
      // y[i] = 9 * sqrt(a*x[i] + y[i]) * a * x[i];
      // x[i] = y[i] + 0.001 * x[i];
      float num = x[i];
      y[i] = x[i] + 0.001f * num;
    }
  }
  
}

int main(void)
{
  int N = 1<<27;
  std::cout << N;
  float *x, *y, *d_x, *d_y;
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));

  hipMalloc(&d_x, N*sizeof(float)); 
  hipMalloc(&d_y, N*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  std::chrono::time_point<std::chrono::system_clock> start, end;
  start = std::chrono::system_clock::now();

  // Perform SAXPY on 1M elements
  saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);

  hipDeviceSynchronize();

  // measure elapsed kernal runtime
  end = std::chrono::system_clock::now();
  std::chrono::duration<double> elapsed_seconds = end - start;
  std::time_t end_time = std::chrono::system_clock::to_time_t(end);
  std::cout << "Elapsed Time: " << elapsed_seconds.count() << "s\n";

  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = max(maxError, abs(y[i]-4.0f));
  printf("Max error: %f\n", maxError);

  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);
}