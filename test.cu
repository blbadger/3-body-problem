#include <stdio.h>
#include <iostream>
#include <chrono>
#include <fstream>
#include <iterator>
#include <fstream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  for (int j=0; j < 1000; j++){
    if (i < n) {
      y[i] = 20*i;
    }
  }
}

// #define SHIFT_AMOUNT 16 // 2^16 = 65536
// #define SHIFT_MASK ((1 << SHIFT_AMOUNT) - 1) // 65535 (all LSB set, all MSB clear)

int main(void) {
  int shift_amount = 28;
  int shift_mask = (1 << shift_amount) - 1;
  int price = -(1 << shift_amount);
  printf ("price is %d\n", price );
  price = price * 3.2435869;
  printf ("price is %d\n", price );
  printf ("price is %d\n", price  >> shift_amount);
  printf ("price fraction is %d\n", price & shift_mask);
  printf("price fraction in decimal is %.10g", ((double)(price & shift_mask) / (1 << shift_amount)));
}

extern "C" {
float* save_arr()
  {
    int N = 1<<20;
    float *x, *y, *d_x, *d_y;
    x = (float*)malloc(N*sizeof(float));
    y = (float*)malloc(N*sizeof(float));

    hipMalloc(&d_x, N*sizeof(float)); 
    hipMalloc(&d_y, N*sizeof(float));

    for (int i = 0; i < N; i++) {
      x[i] = 1.0f;
      y[i] = 2.0f;
    }

    hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

    std::chrono::time_point<std::chrono::system_clock> start, end;
    start = std::chrono::system_clock::now();

    // Perform SAXPY on 1M elements
    saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);

    hipDeviceSynchronize();

    // measure elapsed kernal runtime
    end = std::chrono::system_clock::now();
    std::chrono::duration<double> elapsed_seconds = end - start;
    std::time_t end_time = std::chrono::system_clock::to_time_t(end);
    std::cout << "Elapsed Time: " << elapsed_seconds.count() << "s\n";

    hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
    free(x);
    return y;
  }
}