
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <chrono>

// kernal declaration
__global__
void divergence(int n, 
              int steps,
              double delta_t,
              bool *still_together,
              bool *not_diverged,
              int *times,
              double m_1, double m_2, double m_3,
              double critical_distance,
              double *p1_x, double *p1_y, double *p1_z, 
              double *p2_x, double *p2_y, double *p2_z, 
              double *p3_x, double *p3_y, double *p3_z, 
              double *p1_prime_x, double *p1_prime_y, double *p1_prime_z, 
              double *p2_prime_x, double *p2_prime_y, double *p2_prime_z, 
              double *p3_prime_x, double *p3_prime_y, double *p3_prime_z, 
              double *dv_1_x, double *dv_1_y, double *dv_1_z,
              double *dv_2_x, double *dv_2_y, double *dv_2_z,
              double *dv_3_x, double *dv_3_y, double *dv_3_z,
              double *dv_1pr_x, double *dv_1pr_y, double *dv_1pr_z,
              double *dv_2pr_x, double *dv_2pr_y, double *dv_2pr_z,
              double *dv_3pr_x, double *dv_3pr_y, double *dv_3pr_z,
              double *v1_x, double *v1_y, double *v1_z,
              double *v2_x, double *v2_y, double *v2_z,
              double *v3_x, double *v3_y, double *v3_z,
              double *v1_prime_x, double *v1_prime_y, double *v1_prime_z,
              double *v2_prime_x, double *v2_prime_y, double *v2_prime_z,
              double *v3_prime_x, double *v3_prime_y, double *v3_prime_z,
              double *nv1_x, double *nv1_y, double *nv1_z,
              double *nv2_x, double *nv2_y, double *nv2_z,
              double *nv3_x, double *nv3_y, double *nv3_z,
              double *nv1_prime_x, double *nv1_prime_y, double *nv1_prime_z,
              double *nv2_prime_x, double *nv2_prime_y, double *nv2_prime_z,
              double *nv3_prime_x, double *nv3_prime_y, double *nv3_prime_z
              )
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  for (int j=0; j < steps; j++) {
    if (i < n and still_together[i]){
      // compute accelerations
      dv_1_x[i] = -9.8 * m_2 * (p1_x[i] - p2_x[i]) / (sqrt((p1_x[i] - p2_x[i])*(p1_x[i] - p2_x[i]) + (p1_y[i] - p2_y[i])*(p1_y[i] - p2_y[i]) + (p1_z[i] - p2_z[i])*(p1_z[i] - p2_z[i]))*sqrt((p1_x[i] - p2_x[i])*(p1_x[i] - p2_x[i]) + (p1_y[i] - p2_y[i])*(p1_y[i] - p2_y[i]) + (p1_z[i] - p2_z[i])*(p1_z[i] - p2_z[i]))*sqrt((p1_x[i] - p2_x[i])*(p1_x[i] - p2_x[i]) + (p1_y[i] - p2_y[i])*(p1_y[i] - p2_y[i]) + (p1_z[i] - p2_z[i])*(p1_z[i] - p2_z[i]))) -9.8 * m_3 * (p1_x[i] - p3_x[i]) / (sqrt((p1_x[i] - p3_x[i])*(p1_x[i] - p3_x[i]) + (p1_y[i] - p3_y[i])*(p1_y[i] - p3_y[i]) + (p1_z[i] - p3_z[i])*(p1_z[i] - p3_z[i]))*sqrt((p1_x[i] - p3_x[i])*(p1_x[i] - p3_x[i]) + (p1_y[i] - p3_y[i])*(p1_y[i] - p3_y[i]) + (p1_z[i] - p3_z[i])*(p1_z[i] - p3_z[i]))*sqrt((p1_x[i] - p3_x[i])*(p1_x[i] - p3_x[i]) + (p1_y[i] - p3_y[i])*(p1_y[i] - p3_y[i]) + (p1_z[i] - p3_z[i])*(p1_z[i] - p3_z[i])));
      dv_1_y[i] = -9.8 * m_2 * (p1_y[i] - p2_y[i]) / (sqrt((p1_x[i] - p2_x[i])*(p1_x[i] - p2_x[i]) + (p1_y[i] - p2_y[i])*(p1_y[i] - p2_y[i]) + (p1_z[i] - p2_z[i])*(p1_z[i] - p2_z[i]))*sqrt((p1_x[i] - p2_x[i])*(p1_x[i] - p2_x[i]) + (p1_y[i] - p2_y[i])*(p1_y[i] - p2_y[i]) + (p1_z[i] - p2_z[i])*(p1_z[i] - p2_z[i]))*sqrt((p1_x[i] - p2_x[i])*(p1_x[i] - p2_x[i]) + (p1_y[i] - p2_y[i])*(p1_y[i] - p2_y[i]) + (p1_z[i] - p2_z[i])*(p1_z[i] - p2_z[i]))) -9.8 * m_3 * (p1_y[i] - p3_y[i]) / (sqrt((p1_x[i] - p3_x[i])*(p1_x[i] - p3_x[i]) + (p1_y[i] - p3_y[i])*(p1_y[i] - p3_y[i]) + (p1_z[i] - p3_z[i])*(p1_z[i] - p3_z[i]))*sqrt((p1_x[i] - p3_x[i])*(p1_x[i] - p3_x[i]) + (p1_y[i] - p3_y[i])*(p1_y[i] - p3_y[i]) + (p1_z[i] - p3_z[i])*(p1_z[i] - p3_z[i]))*sqrt((p1_x[i] - p3_x[i])*(p1_x[i] - p3_x[i]) + (p1_y[i] - p3_y[i])*(p1_y[i] - p3_y[i]) + (p1_z[i] - p3_z[i])*(p1_z[i] - p3_z[i])));
      dv_1_z[i] = -9.8 * m_2 * (p1_z[i] - p2_z[i]) / (sqrt((p1_x[i] - p2_x[i])*(p1_x[i] - p2_x[i]) + (p1_y[i] - p2_y[i])*(p1_y[i] - p2_y[i]) + (p1_z[i] - p2_z[i])*(p1_z[i] - p2_z[i]))*sqrt((p1_x[i] - p2_x[i])*(p1_x[i] - p2_x[i]) + (p1_y[i] - p2_y[i])*(p1_y[i] - p2_y[i]) + (p1_z[i] - p2_z[i])*(p1_z[i] - p2_z[i]))*sqrt((p1_x[i] - p2_x[i])*(p1_x[i] - p2_x[i]) + (p1_y[i] - p2_y[i])*(p1_y[i] - p2_y[i]) + (p1_z[i] - p2_z[i])*(p1_z[i] - p2_z[i]))) -9.8 * m_3 * (p1_z[i] - p3_z[i]) / (sqrt((p1_x[i] - p3_x[i])*(p1_x[i] - p3_x[i]) + (p1_y[i] - p3_y[i])*(p1_y[i] - p3_y[i]) + (p1_z[i] - p3_z[i])*(p1_z[i] - p3_z[i]))*sqrt((p1_x[i] - p3_x[i])*(p1_x[i] - p3_x[i]) + (p1_y[i] - p3_y[i])*(p1_y[i] - p3_y[i]) + (p1_z[i] - p3_z[i])*(p1_z[i] - p3_z[i]))*sqrt((p1_x[i] - p3_x[i])*(p1_x[i] - p3_x[i]) + (p1_y[i] - p3_y[i])*(p1_y[i] - p3_y[i]) + (p1_z[i] - p3_z[i])*(p1_z[i] - p3_z[i])));
      
      dv_2_x[i] = -9.8 * m_3 * (p2_x[i] - p3_x[i]) / (sqrt((p2_x[i] - p3_x[i])*(p2_x[i] - p3_x[i]) + (p2_y[i] - p3_y[i])*(p2_y[i] - p3_y[i]) + (p2_z[i] - p3_z[i])*(p2_z[i] - p3_z[i]))*sqrt((p2_x[i] - p3_x[i])*(p2_x[i] - p3_x[i]) + (p2_y[i] - p3_y[i])*(p2_y[i] - p3_y[i]) + (p2_z[i] - p3_z[i])*(p2_z[i] - p3_z[i]))*sqrt((p2_x[i] - p3_x[i])*(p2_x[i] - p3_x[i]) + (p2_y[i] - p3_y[i])*(p2_y[i] - p3_y[i]) + (p2_z[i] - p3_z[i])*(p2_z[i] - p3_z[i]))) -9.8 * m_1 * (p2_x[i] - p1_x[i]) / (sqrt((p2_x[i] - p1_x[i])*(p2_x[i] - p1_x[i]) + (p2_y[i] - p1_y[i])*(p2_y[i] - p1_y[i]) + (p2_z[i] - p1_z[i])*(p2_z[i] - p1_z[i]))*sqrt((p2_x[i] - p1_x[i])*(p2_x[i] - p1_x[i]) + (p2_y[i] - p1_y[i])*(p2_y[i] - p1_y[i]) + (p2_z[i] - p1_z[i])*(p2_z[i] - p1_z[i]))*sqrt((p2_x[i] - p1_x[i])*(p2_x[i] - p1_x[i]) + (p2_y[i] - p1_y[i])*(p2_y[i] - p1_y[i]) + (p2_z[i] - p1_z[i])*(p2_z[i] - p1_z[i])));
      dv_2_y[i] = -9.8 * m_3 * (p2_y[i] - p3_y[i]) / (sqrt((p2_x[i] - p3_x[i])*(p2_x[i] - p3_x[i]) + (p2_y[i] - p3_y[i])*(p2_y[i] - p3_y[i]) + (p2_z[i] - p3_z[i])*(p2_z[i] - p3_z[i]))*sqrt((p2_x[i] - p3_x[i])*(p2_x[i] - p3_x[i]) + (p2_y[i] - p3_y[i])*(p2_y[i] - p3_y[i]) + (p2_z[i] - p3_z[i])*(p2_z[i] - p3_z[i]))*sqrt((p2_x[i] - p3_x[i])*(p2_x[i] - p3_x[i]) + (p2_y[i] - p3_y[i])*(p2_y[i] - p3_y[i]) + (p2_z[i] - p3_z[i])*(p2_z[i] - p3_z[i]))) -9.8 * m_1 * (p2_y[i] - p1_y[i]) / (sqrt((p2_x[i] - p1_x[i])*(p2_x[i] - p1_x[i]) + (p2_y[i] - p1_y[i])*(p2_y[i] - p1_y[i]) + (p2_z[i] - p1_z[i])*(p2_z[i] - p1_z[i]))*sqrt((p2_x[i] - p1_x[i])*(p2_x[i] - p1_x[i]) + (p2_y[i] - p1_y[i])*(p2_y[i] - p1_y[i]) + (p2_z[i] - p1_z[i])*(p2_z[i] - p1_z[i]))*sqrt((p2_x[i] - p1_x[i])*(p2_x[i] - p1_x[i]) + (p2_y[i] - p1_y[i])*(p2_y[i] - p1_y[i]) + (p2_z[i] - p1_z[i])*(p2_z[i] - p1_z[i])));
      dv_2_z[i] = -9.8 * m_3 * (p2_z[i] - p3_z[i]) / (sqrt((p2_x[i] - p3_x[i])*(p2_x[i] - p3_x[i]) + (p2_y[i] - p3_y[i])*(p2_y[i] - p3_y[i]) + (p2_z[i] - p3_z[i])*(p2_z[i] - p3_z[i]))*sqrt((p2_x[i] - p3_x[i])*(p2_x[i] - p3_x[i]) + (p2_y[i] - p3_y[i])*(p2_y[i] - p3_y[i]) + (p2_z[i] - p3_z[i])*(p2_z[i] - p3_z[i]))*sqrt((p2_x[i] - p3_x[i])*(p2_x[i] - p3_x[i]) + (p2_y[i] - p3_y[i])*(p2_y[i] - p3_y[i]) + (p2_z[i] - p3_z[i])*(p2_z[i] - p3_z[i]))) -9.8 * m_1 * (p2_z[i] - p1_z[i]) / (sqrt((p2_x[i] - p1_x[i])*(p2_x[i] - p1_x[i]) + (p2_y[i] - p1_y[i])*(p2_y[i] - p1_y[i]) + (p2_z[i] - p1_z[i])*(p2_z[i] - p1_z[i]))*sqrt((p2_x[i] - p1_x[i])*(p2_x[i] - p1_x[i]) + (p2_y[i] - p1_y[i])*(p2_y[i] - p1_y[i]) + (p2_z[i] - p1_z[i])*(p2_z[i] - p1_z[i]))*sqrt((p2_x[i] - p1_x[i])*(p2_x[i] - p1_x[i]) + (p2_y[i] - p1_y[i])*(p2_y[i] - p1_y[i]) + (p2_z[i] - p1_z[i])*(p2_z[i] - p1_z[i])));

      dv_3_x[i] = -9.8 * m_1 * (p3_x[i] - p1_x[i]) / (sqrt((p3_x[i] - p1_x[i])*(p3_x[i] - p1_x[i]) + (p3_y[i] - p1_y[i])*(p3_y[i] - p1_y[i]) + (p3_z[i] - p1_z[i])*(p3_z[i] - p1_z[i]))*sqrt((p3_x[i] - p1_x[i])*(p3_x[i] - p1_x[i]) + (p3_y[i] - p1_y[i])*(p3_y[i] - p1_y[i]) + (p3_z[i] - p1_z[i])*(p3_z[i] - p1_z[i]))*sqrt((p3_x[i] - p1_x[i])*(p3_x[i] - p1_x[i]) + (p3_y[i] - p1_y[i])*(p3_y[i] - p1_y[i]) + (p3_z[i] - p1_z[i])*(p3_z[i] - p1_z[i]))) -9.8 * m_2 * (p3_x[i] - p2_x[i]) / (sqrt((p3_x[i] - p2_x[i])*(p3_x[i] - p2_x[i]) + (p3_y[i] - p2_y[i])*(p3_y[i] - p2_y[i]) + (p3_z[i] - p2_z[i])*(p3_z[i] - p2_z[i]))*sqrt((p3_x[i] - p2_x[i])*(p3_x[i] - p2_x[i]) + (p3_y[i] - p2_y[i])*(p3_y[i] - p2_y[i]) + (p3_z[i] - p2_z[i])*(p3_z[i] - p2_z[i]))*sqrt((p3_x[i] - p2_x[i])*(p3_x[i] - p2_x[i]) + (p3_y[i] - p2_y[i])*(p3_y[i] - p2_y[i]) + (p3_z[i] - p2_z[i])*(p3_z[i] - p2_z[i])));
      dv_3_y[i] = -9.8 * m_1 * (p3_y[i] - p1_y[i]) / (sqrt((p3_x[i] - p1_x[i])*(p3_x[i] - p1_x[i]) + (p3_y[i] - p1_y[i])*(p3_y[i] - p1_y[i]) + (p3_z[i] - p1_z[i])*(p3_z[i] - p1_z[i]))*sqrt((p3_x[i] - p1_x[i])*(p3_x[i] - p1_x[i]) + (p3_y[i] - p1_y[i])*(p3_y[i] - p1_y[i]) + (p3_z[i] - p1_z[i])*(p3_z[i] - p1_z[i]))*sqrt((p3_x[i] - p1_x[i])*(p3_x[i] - p1_x[i]) + (p3_y[i] - p1_y[i])*(p3_y[i] - p1_y[i]) + (p3_z[i] - p1_z[i])*(p3_z[i] - p1_z[i]))) -9.8 * m_2 * (p3_y[i] - p2_y[i]) / (sqrt((p3_x[i] - p2_x[i])*(p3_x[i] - p2_x[i]) + (p3_y[i] - p2_y[i])*(p3_y[i] - p2_y[i]) + (p3_z[i] - p2_z[i])*(p3_z[i] - p2_z[i]))*sqrt((p3_x[i] - p2_x[i])*(p3_x[i] - p2_x[i]) + (p3_y[i] - p2_y[i])*(p3_y[i] - p2_y[i]) + (p3_z[i] - p2_z[i])*(p3_z[i] - p2_z[i]))*sqrt((p3_x[i] - p2_x[i])*(p3_x[i] - p2_x[i]) + (p3_y[i] - p2_y[i])*(p3_y[i] - p2_y[i]) + (p3_z[i] - p2_z[i])*(p3_z[i] - p2_z[i])));
      dv_3_z[i] = -9.8 * m_1 * (p3_z[i] - p1_z[i]) / (sqrt((p3_x[i] - p1_x[i])*(p3_x[i] - p1_x[i]) + (p3_y[i] - p1_y[i])*(p3_y[i] - p1_y[i]) + (p3_z[i] - p1_z[i])*(p3_z[i] - p1_z[i]))*sqrt((p3_x[i] - p1_x[i])*(p3_x[i] - p1_x[i]) + (p3_y[i] - p1_y[i])*(p3_y[i] - p1_y[i]) + (p3_z[i] - p1_z[i])*(p3_z[i] - p1_z[i]))*sqrt((p3_x[i] - p1_x[i])*(p3_x[i] - p1_x[i]) + (p3_y[i] - p1_y[i])*(p3_y[i] - p1_y[i]) + (p3_z[i] - p1_z[i])*(p3_z[i] - p1_z[i]))) -9.8 * m_2 * (p3_z[i] - p2_z[i]) / (sqrt((p3_x[i] - p2_x[i])*(p3_x[i] - p2_x[i]) + (p3_y[i] - p2_y[i])*(p3_y[i] - p2_y[i]) + (p3_z[i] - p2_z[i])*(p3_z[i] - p2_z[i]))*sqrt((p3_x[i] - p2_x[i])*(p3_x[i] - p2_x[i]) + (p3_y[i] - p2_y[i])*(p3_y[i] - p2_y[i]) + (p3_z[i] - p2_z[i])*(p3_z[i] - p2_z[i]))*sqrt((p3_x[i] - p2_x[i])*(p3_x[i] - p2_x[i]) + (p3_y[i] - p2_y[i])*(p3_y[i] - p2_y[i]) + (p3_z[i] - p2_z[i])*(p3_z[i] - p2_z[i])));

      dv_1pr_x[i] = -9.8 * m_2 * (p1_prime_x[i] - p2_prime_x[i]) / (sqrt((p1_prime_x[i] - p2_prime_x[i])*(p1_prime_x[i] - p2_prime_x[i]) + (p1_prime_y[i] - p2_prime_y[i])*(p1_prime_y[i] - p2_prime_y[i]) + (p1_prime_z[i] - p2_prime_z[i])*(p1_prime_z[i] - p2_prime_z[i]))*sqrt((p1_prime_x[i] - p2_prime_x[i])*(p1_prime_x[i] - p2_prime_x[i]) + (p1_prime_y[i] - p2_prime_y[i])*(p1_prime_y[i] - p2_prime_y[i]) + (p1_prime_z[i] - p2_prime_z[i])*(p1_prime_z[i] - p2_prime_z[i]))*sqrt((p1_prime_x[i] - p2_prime_x[i])*(p1_prime_x[i] - p2_prime_x[i]) + (p1_prime_y[i] - p2_prime_y[i])*(p1_prime_y[i] - p2_prime_y[i]) + (p1_prime_z[i] - p2_prime_z[i])*(p1_prime_z[i] - p2_prime_z[i]))) -9.8 * m_3 * (p1_prime_x[i] - p3_prime_x[i]) / (sqrt((p1_prime_x[i] - p3_prime_x[i])*(p1_prime_x[i] - p3_prime_x[i]) + (p1_prime_y[i] - p3_prime_y[i])*(p1_prime_y[i] - p3_prime_y[i]) + (p1_prime_z[i] - p3_prime_z[i])*(p1_prime_z[i] - p3_prime_z[i]))*sqrt((p1_prime_x[i] - p3_prime_x[i])*(p1_prime_x[i] - p3_prime_x[i]) + (p1_prime_y[i] - p3_prime_y[i])*(p1_prime_y[i] - p3_prime_y[i]) + (p1_prime_z[i] - p3_prime_z[i])*(p1_prime_z[i] - p3_prime_z[i]))*sqrt((p1_prime_x[i] - p3_prime_x[i])*(p1_prime_x[i] - p3_prime_x[i]) + (p1_prime_y[i] - p3_prime_y[i])*(p1_prime_y[i] - p3_prime_y[i]) + (p1_prime_z[i] - p3_prime_z[i])*(p1_prime_z[i] - p3_prime_z[i])));
      dv_1pr_y[i] = -9.8 * m_2 * (p1_prime_y[i] - p2_prime_y[i]) / (sqrt((p1_prime_x[i] - p2_prime_x[i])*(p1_prime_x[i] - p2_prime_x[i]) + (p1_prime_y[i] - p2_prime_y[i])*(p1_prime_y[i] - p2_prime_y[i]) + (p1_prime_z[i] - p2_prime_z[i])*(p1_prime_z[i] - p2_prime_z[i]))*sqrt((p1_prime_x[i] - p2_prime_x[i])*(p1_prime_x[i] - p2_prime_x[i]) + (p1_prime_y[i] - p2_prime_y[i])*(p1_prime_y[i] - p2_prime_y[i]) + (p1_prime_z[i] - p2_prime_z[i])*(p1_prime_z[i] - p2_prime_z[i]))*sqrt((p1_prime_x[i] - p2_prime_x[i])*(p1_prime_x[i] - p2_prime_x[i]) + (p1_prime_y[i] - p2_prime_y[i])*(p1_prime_y[i] - p2_prime_y[i]) + (p1_prime_z[i] - p2_prime_z[i])*(p1_prime_z[i] - p2_prime_z[i]))) -9.8 * m_3 * (p1_prime_y[i] - p3_prime_y[i]) / (sqrt((p1_prime_x[i] - p3_prime_x[i])*(p1_prime_x[i] - p3_prime_x[i]) + (p1_prime_y[i] - p3_prime_y[i])*(p1_prime_y[i] - p3_prime_y[i]) + (p1_prime_z[i] - p3_prime_z[i])*(p1_prime_z[i] - p3_prime_z[i]))*sqrt((p1_prime_x[i] - p3_prime_x[i])*(p1_prime_x[i] - p3_prime_x[i]) + (p1_prime_y[i] - p3_prime_y[i])*(p1_prime_y[i] - p3_prime_y[i]) + (p1_prime_z[i] - p3_prime_z[i])*(p1_prime_z[i] - p3_prime_z[i]))*sqrt((p1_prime_x[i] - p3_prime_x[i])*(p1_prime_x[i] - p3_prime_x[i]) + (p1_prime_y[i] - p3_prime_y[i])*(p1_prime_y[i] - p3_prime_y[i]) + (p1_prime_z[i] - p3_prime_z[i])*(p1_prime_z[i] - p3_prime_z[i])));
      dv_1pr_z[i] = -9.8 * m_2 * (p1_prime_z[i] - p2_prime_z[i]) / (sqrt((p1_prime_x[i] - p2_prime_x[i])*(p1_prime_x[i] - p2_prime_x[i]) + (p1_prime_y[i] - p2_prime_y[i])*(p1_prime_y[i] - p2_prime_y[i]) + (p1_prime_z[i] - p2_prime_z[i])*(p1_prime_z[i] - p2_prime_z[i]))*sqrt((p1_prime_x[i] - p2_prime_x[i])*(p1_prime_x[i] - p2_prime_x[i]) + (p1_prime_y[i] - p2_prime_y[i])*(p1_prime_y[i] - p2_prime_y[i]) + (p1_prime_z[i] - p2_prime_z[i])*(p1_prime_z[i] - p2_prime_z[i]))*sqrt((p1_prime_x[i] - p2_prime_x[i])*(p1_prime_x[i] - p2_prime_x[i]) + (p1_prime_y[i] - p2_prime_y[i])*(p1_prime_y[i] - p2_prime_y[i]) + (p1_prime_z[i] - p2_prime_z[i])*(p1_prime_z[i] - p2_prime_z[i]))) -9.8 * m_3 * (p1_prime_z[i] - p3_prime_z[i]) / (sqrt((p1_prime_x[i] - p3_prime_x[i])*(p1_prime_x[i] - p3_prime_x[i]) + (p1_prime_y[i] - p3_prime_y[i])*(p1_prime_y[i] - p3_prime_y[i]) + (p1_prime_z[i] - p3_prime_z[i])*(p1_prime_z[i] - p3_prime_z[i]))*sqrt((p1_prime_x[i] - p3_prime_x[i])*(p1_prime_x[i] - p3_prime_x[i]) + (p1_prime_y[i] - p3_prime_y[i])*(p1_prime_y[i] - p3_prime_y[i]) + (p1_prime_z[i] - p3_prime_z[i])*(p1_prime_z[i] - p3_prime_z[i]))*sqrt((p1_prime_x[i] - p3_prime_x[i])*(p1_prime_x[i] - p3_prime_x[i]) + (p1_prime_y[i] - p3_prime_y[i])*(p1_prime_y[i] - p3_prime_y[i]) + (p1_prime_z[i] - p3_prime_z[i])*(p1_prime_z[i] - p3_prime_z[i])));
      
      dv_2pr_x[i] = -9.8 * m_3 * (p2_prime_x[i] - p3_prime_x[i]) / (sqrt((p2_prime_x[i] - p3_prime_x[i])*(p2_prime_x[i] - p3_prime_x[i]) + (p2_prime_y[i] - p3_prime_y[i])*(p2_prime_y[i] - p3_prime_y[i]) + (p2_prime_z[i] - p3_prime_z[i])*(p2_prime_z[i] - p3_prime_z[i]))*sqrt((p2_prime_x[i] - p3_prime_x[i])*(p2_prime_x[i] - p3_prime_x[i]) + (p2_prime_y[i] - p3_prime_y[i])*(p2_prime_y[i] - p3_prime_y[i]) + (p2_prime_z[i] - p3_prime_z[i])*(p2_prime_z[i] - p3_prime_z[i]))*sqrt((p2_prime_x[i] - p3_prime_x[i])*(p2_prime_x[i] - p3_prime_x[i]) + (p2_prime_y[i] - p3_prime_y[i])*(p2_prime_y[i] - p3_prime_y[i]) + (p2_prime_z[i] - p3_prime_z[i])*(p2_prime_z[i] - p3_prime_z[i]))) -9.8 * m_1 * (p2_prime_x[i] - p1_prime_x[i]) / (sqrt((p2_prime_x[i] - p1_prime_x[i])*(p2_prime_x[i] - p1_prime_x[i]) + (p2_prime_y[i] - p1_prime_y[i])*(p2_prime_y[i] - p1_prime_y[i]) + (p2_prime_z[i] - p1_prime_z[i])*(p2_prime_z[i] - p1_prime_z[i]))*sqrt((p2_prime_x[i] - p1_prime_x[i])*(p2_prime_x[i] - p1_prime_x[i]) + (p2_prime_y[i] - p1_prime_y[i])*(p2_prime_y[i] - p1_prime_y[i]) + (p2_prime_z[i] - p1_prime_z[i])*(p2_prime_z[i] - p1_prime_z[i]))*sqrt((p2_prime_x[i] - p1_prime_x[i])*(p2_prime_x[i] - p1_prime_x[i]) + (p2_prime_y[i] - p1_prime_y[i])*(p2_prime_y[i] - p1_prime_y[i]) + (p2_prime_z[i] - p1_prime_z[i])*(p2_prime_z[i] - p1_prime_z[i])));
      dv_2pr_y[i] = -9.8 * m_3 * (p2_prime_y[i] - p3_prime_y[i]) / (sqrt((p2_prime_x[i] - p3_prime_x[i])*(p2_prime_x[i] - p3_prime_x[i]) + (p2_prime_y[i] - p3_prime_y[i])*(p2_prime_y[i] - p3_prime_y[i]) + (p2_prime_z[i] - p3_prime_z[i])*(p2_prime_z[i] - p3_prime_z[i]))*sqrt((p2_prime_x[i] - p3_prime_x[i])*(p2_prime_x[i] - p3_prime_x[i]) + (p2_prime_y[i] - p3_prime_y[i])*(p2_prime_y[i] - p3_prime_y[i]) + (p2_prime_z[i] - p3_prime_z[i])*(p2_prime_z[i] - p3_prime_z[i]))*sqrt((p2_prime_x[i] - p3_prime_x[i])*(p2_prime_x[i] - p3_prime_x[i]) + (p2_prime_y[i] - p3_prime_y[i])*(p2_prime_y[i] - p3_prime_y[i]) + (p2_prime_z[i] - p3_prime_z[i])*(p2_prime_z[i] - p3_prime_z[i]))) -9.8 * m_1 * (p2_prime_y[i] - p1_prime_y[i]) / (sqrt((p2_prime_x[i] - p1_prime_x[i])*(p2_prime_x[i] - p1_prime_x[i]) + (p2_prime_y[i] - p1_prime_y[i])*(p2_prime_y[i] - p1_prime_y[i]) + (p2_prime_z[i] - p1_prime_z[i])*(p2_prime_z[i] - p1_prime_z[i]))*sqrt((p2_prime_x[i] - p1_prime_x[i])*(p2_prime_x[i] - p1_prime_x[i]) + (p2_prime_y[i] - p1_prime_y[i])*(p2_prime_y[i] - p1_prime_y[i]) + (p2_prime_z[i] - p1_prime_z[i])*(p2_prime_z[i] - p1_prime_z[i]))*sqrt((p2_prime_x[i] - p1_prime_x[i])*(p2_prime_x[i] - p1_prime_x[i]) + (p2_prime_y[i] - p1_prime_y[i])*(p2_prime_y[i] - p1_prime_y[i]) + (p2_prime_z[i] - p1_prime_z[i])*(p2_prime_z[i] - p1_prime_z[i])));
      dv_2pr_z[i] = -9.8 * m_3 * (p2_prime_z[i] - p3_prime_z[i]) / (sqrt((p2_prime_x[i] - p3_prime_x[i])*(p2_prime_x[i] - p3_prime_x[i]) + (p2_prime_y[i] - p3_prime_y[i])*(p2_prime_y[i] - p3_prime_y[i]) + (p2_prime_z[i] - p3_prime_z[i])*(p2_prime_z[i] - p3_prime_z[i]))*sqrt((p2_prime_x[i] - p3_prime_x[i])*(p2_prime_x[i] - p3_prime_x[i]) + (p2_prime_y[i] - p3_prime_y[i])*(p2_prime_y[i] - p3_prime_y[i]) + (p2_prime_z[i] - p3_prime_z[i])*(p2_prime_z[i] - p3_prime_z[i]))*sqrt((p2_prime_x[i] - p3_prime_x[i])*(p2_prime_x[i] - p3_prime_x[i]) + (p2_prime_y[i] - p3_prime_y[i])*(p2_prime_y[i] - p3_prime_y[i]) + (p2_prime_z[i] - p3_prime_z[i])*(p2_prime_z[i] - p3_prime_z[i]))) -9.8 * m_1 * (p2_prime_z[i] - p1_prime_z[i]) / (sqrt((p2_prime_x[i] - p1_prime_x[i])*(p2_prime_x[i] - p1_prime_x[i]) + (p2_prime_y[i] - p1_prime_y[i])*(p2_prime_y[i] - p1_prime_y[i]) + (p2_prime_z[i] - p1_prime_z[i])*(p2_prime_z[i] - p1_prime_z[i]))*sqrt((p2_prime_x[i] - p1_prime_x[i])*(p2_prime_x[i] - p1_prime_x[i]) + (p2_prime_y[i] - p1_prime_y[i])*(p2_prime_y[i] - p1_prime_y[i]) + (p2_prime_z[i] - p1_prime_z[i])*(p2_prime_z[i] - p1_prime_z[i]))*sqrt((p2_prime_x[i] - p1_prime_x[i])*(p2_prime_x[i] - p1_prime_x[i]) + (p2_prime_y[i] - p1_prime_y[i])*(p2_prime_y[i] - p1_prime_y[i]) + (p2_prime_z[i] - p1_prime_z[i])*(p2_prime_z[i] - p1_prime_z[i])));

      dv_3pr_x[i] = -9.8 * m_1 * (p3_prime_x[i] - p1_prime_x[i]) / (sqrt((p3_prime_x[i] - p1_prime_x[i])*(p3_prime_x[i] - p1_prime_x[i]) + (p3_prime_y[i] - p1_prime_y[i])*(p3_prime_y[i] - p1_prime_y[i]) + (p3_prime_z[i] - p1_prime_z[i])*(p3_prime_z[i] - p1_prime_z[i]))*sqrt((p3_prime_x[i] - p1_prime_x[i])*(p3_prime_x[i] - p1_prime_x[i]) + (p3_prime_y[i] - p1_prime_y[i])*(p3_prime_y[i] - p1_prime_y[i]) + (p3_prime_z[i] - p1_prime_z[i])*(p3_prime_z[i] - p1_prime_z[i]))*sqrt((p3_prime_x[i] - p1_prime_x[i])*(p3_prime_x[i] - p1_prime_x[i]) + (p3_prime_y[i] - p1_prime_y[i])*(p3_prime_y[i] - p1_prime_y[i]) + (p3_prime_z[i] - p1_prime_z[i])*(p3_prime_z[i] - p1_prime_z[i]))) -9.8 * m_2 * (p3_prime_x[i] - p2_prime_x[i]) / (sqrt((p3_prime_x[i] - p2_prime_x[i])*(p3_prime_x[i] - p2_prime_x[i]) + (p3_prime_y[i] - p2_prime_y[i])*(p3_prime_y[i] - p2_prime_y[i]) + (p3_prime_z[i] - p2_prime_z[i])*(p3_prime_z[i] - p2_prime_z[i]))*sqrt((p3_prime_x[i] - p2_prime_x[i])*(p3_prime_x[i] - p2_prime_x[i]) + (p3_prime_y[i] - p2_prime_y[i])*(p3_prime_y[i] - p2_prime_y[i]) + (p3_prime_z[i] - p2_prime_z[i])*(p3_prime_z[i] - p2_prime_z[i]))*sqrt((p3_prime_x[i] - p2_prime_x[i])*(p3_prime_x[i] - p2_prime_x[i]) + (p3_prime_y[i] - p2_prime_y[i])*(p3_prime_y[i] - p2_prime_y[i]) + (p3_prime_z[i] - p2_prime_z[i])*(p3_prime_z[i] - p2_prime_z[i])));
      dv_3pr_y[i] = -9.8 * m_1 * (p3_prime_y[i] - p1_prime_y[i]) / (sqrt((p3_prime_x[i] - p1_prime_x[i])*(p3_prime_x[i] - p1_prime_x[i]) + (p3_prime_y[i] - p1_prime_y[i])*(p3_prime_y[i] - p1_prime_y[i]) + (p3_prime_z[i] - p1_prime_z[i])*(p3_prime_z[i] - p1_prime_z[i]))*sqrt((p3_prime_x[i] - p1_prime_x[i])*(p3_prime_x[i] - p1_prime_x[i]) + (p3_prime_y[i] - p1_prime_y[i])*(p3_prime_y[i] - p1_prime_y[i]) + (p3_prime_z[i] - p1_prime_z[i])*(p3_prime_z[i] - p1_prime_z[i]))*sqrt((p3_prime_x[i] - p1_prime_x[i])*(p3_prime_x[i] - p1_prime_x[i]) + (p3_prime_y[i] - p1_prime_y[i])*(p3_prime_y[i] - p1_prime_y[i]) + (p3_prime_z[i] - p1_prime_z[i])*(p3_prime_z[i] - p1_prime_z[i]))) -9.8 * m_2 * (p3_prime_y[i] - p2_prime_y[i]) / (sqrt((p3_prime_x[i] - p2_prime_x[i])*(p3_prime_x[i] - p2_prime_x[i]) + (p3_prime_y[i] - p2_prime_y[i])*(p3_prime_y[i] - p2_prime_y[i]) + (p3_prime_z[i] - p2_prime_z[i])*(p3_prime_z[i] - p2_prime_z[i]))*sqrt((p3_prime_x[i] - p2_prime_x[i])*(p3_prime_x[i] - p2_prime_x[i]) + (p3_prime_y[i] - p2_prime_y[i])*(p3_prime_y[i] - p2_prime_y[i]) + (p3_prime_z[i] - p2_prime_z[i])*(p3_prime_z[i] - p2_prime_z[i]))*sqrt((p3_prime_x[i] - p2_prime_x[i])*(p3_prime_x[i] - p2_prime_x[i]) + (p3_prime_y[i] - p2_prime_y[i])*(p3_prime_y[i] - p2_prime_y[i]) + (p3_prime_z[i] - p2_prime_z[i])*(p3_prime_z[i] - p2_prime_z[i])));
      dv_3pr_z[i] = -9.8 * m_1 * (p3_prime_z[i] - p1_prime_z[i]) / (sqrt((p3_prime_x[i] - p1_prime_x[i])*(p3_prime_x[i] - p1_prime_x[i]) + (p3_prime_y[i] - p1_prime_y[i])*(p3_prime_y[i] - p1_prime_y[i]) + (p3_prime_z[i] - p1_prime_z[i])*(p3_prime_z[i] - p1_prime_z[i]))*sqrt((p3_prime_x[i] - p1_prime_x[i])*(p3_prime_x[i] - p1_prime_x[i]) + (p3_prime_y[i] - p1_prime_y[i])*(p3_prime_y[i] - p1_prime_y[i]) + (p3_prime_z[i] - p1_prime_z[i])*(p3_prime_z[i] - p1_prime_z[i]))*sqrt((p3_prime_x[i] - p1_prime_x[i])*(p3_prime_x[i] - p1_prime_x[i]) + (p3_prime_y[i] - p1_prime_y[i])*(p3_prime_y[i] - p1_prime_y[i]) + (p3_prime_z[i] - p1_prime_z[i])*(p3_prime_z[i] - p1_prime_z[i]))) -9.8 * m_2 * (p3_prime_z[i] - p2_prime_z[i]) / (sqrt((p3_prime_x[i] - p2_prime_x[i])*(p3_prime_x[i] - p2_prime_x[i]) + (p3_prime_y[i] - p2_prime_y[i])*(p3_prime_y[i] - p2_prime_y[i]) + (p3_prime_z[i] - p2_prime_z[i])*(p3_prime_z[i] - p2_prime_z[i]))*sqrt((p3_prime_x[i] - p2_prime_x[i])*(p3_prime_x[i] - p2_prime_x[i]) + (p3_prime_y[i] - p2_prime_y[i])*(p3_prime_y[i] - p2_prime_y[i]) + (p3_prime_z[i] - p2_prime_z[i])*(p3_prime_z[i] - p2_prime_z[i]))*sqrt((p3_prime_x[i] - p2_prime_x[i])*(p3_prime_x[i] - p2_prime_x[i]) + (p3_prime_y[i] - p2_prime_y[i])*(p3_prime_y[i] - p2_prime_y[i]) + (p3_prime_z[i] - p2_prime_z[i])*(p3_prime_z[i] - p2_prime_z[i])));


      // find which trajectories have diverged and increment *times
      not_diverged[i] = (p1_x[i]-p1_prime_x[i])*(p1_x[i]-p1_prime_x[i]) + (p1_y[i]-p1_prime_y[i])*(p1_y[i]-p1_prime_y[i]) + (p1_z[i]-p1_prime_z[i])*(p1_z[i]-p1_prime_z[i]) <= critical_distance*critical_distance;
      still_together[i] = not_diverged[i] & still_together[i]; // bitwise and 
      if (still_together[i] == 1){
        times[i]++;
      };

      // compute new velocities
      nv1_x[i] = v1_x[i] + delta_t * dv_1_x[i];
      nv1_y[i] = v1_y[i] + delta_t * dv_1_y[i];
      nv1_z[i] = v1_z[i] + delta_t * dv_1_z[i];

      nv2_x[i] = v2_x[i] + delta_t * dv_2_x[i];
      nv2_y[i] = v2_y[i] + delta_t * dv_2_y[i];
      nv2_z[i] = v2_z[i] + delta_t * dv_2_z[i];

      nv3_x[i] = v3_x[i] + delta_t * dv_3_x[i];
      nv3_y[i] = v3_y[i] + delta_t * dv_3_y[i];
      nv3_z[i] = v3_z[i] + delta_t * dv_3_z[i];

      nv1_prime_x[i] = v1_prime_x[i] + delta_t * dv_1pr_x[i];
      nv1_prime_y[i] = v1_prime_y[i] + delta_t * dv_1pr_y[i];
      nv1_prime_z[i] = v1_prime_z[i] + delta_t * dv_1pr_z[i];

      nv2_prime_x[i] = v2_prime_x[i] + delta_t * dv_2pr_x[i];
      nv2_prime_y[i] = v2_prime_y[i] + delta_t * dv_2pr_y[i];
      nv2_prime_z[i] = v2_prime_z[i] + delta_t * dv_2pr_z[i];

      nv3_prime_x[i] = v3_prime_x[i] + delta_t * dv_3pr_x[i];
      nv3_prime_y[i] = v3_prime_y[i] + delta_t * dv_3pr_y[i];
      nv3_prime_z[i] = v3_prime_z[i] + delta_t * dv_3pr_z[i];

      // compute positions with current velocities
      p1_x[i] = p1_x[i] + delta_t * v1_x[i];
      p1_y[i] = p1_y[i] + delta_t * v1_y[i];
      p1_z[i] = p1_z[i] + delta_t * v1_z[i];

      p2_x[i] = p2_x[i] + delta_t * v2_x[i];
      p2_y[i] = p2_y[i] + delta_t * v2_y[i];
      p2_z[i] = p2_z[i] + delta_t * v2_z[i];

      p3_x[i] = p3_x[i] + delta_t * v3_x[i];
      p3_y[i] = p3_y[i] + delta_t * v3_y[i];
      p3_z[i] = p3_z[i] + delta_t * v3_z[i];

      p1_prime_x[i] = p1_prime_x[i] + delta_t * v1_prime_x[i];
      p1_prime_y[i] = p1_prime_y[i] + delta_t * v1_prime_y[i];
      p1_prime_z[i] = p1_prime_z[i] + delta_t * v1_prime_z[i];

      p2_prime_x[i] = p2_prime_x[i] + delta_t * v2_prime_x[i];
      p2_prime_y[i] = p2_prime_y[i] + delta_t * v2_prime_y[i];
      p2_prime_z[i] = p2_prime_z[i] + delta_t * v2_prime_z[i];

      p3_prime_x[i] = p3_prime_x[i] + delta_t * v3_prime_x[i];
      p3_prime_y[i] = p3_prime_y[i] + delta_t * v3_prime_y[i];
      p3_prime_z[i] = p3_prime_z[i] + delta_t * v3_prime_z[i];

      // assign new velocities to current velocities
      v1_x[i] = nv1_x[i];
      v1_y[i] = nv1_y[i];
      v1_z[i] = nv1_z[i];

      v2_x[i] = nv2_x[i];
      v2_y[i] = nv2_y[i];
      v2_z[i] = nv2_z[i];

      v3_x[i] = nv3_x[i];
      v3_y[i] = nv3_y[i];
      v3_z[i] = nv3_z[i];

      v1_prime_x[i] = nv1_prime_x[i];
      v1_prime_y[i] = nv1_prime_y[i];
      v1_prime_z[i] = nv1_prime_z[i];

      v2_prime_x[i] = nv2_prime_x[i];
      v2_prime_y[i] = nv2_prime_y[i];
      v2_prime_z[i] = nv2_prime_z[i];

      v3_prime_x[i] = nv3_prime_x[i];
      v3_prime_y[i] = nv3_prime_y[i];
      v3_prime_z[i] = nv3_prime_z[i];
      }
    }
  }

int main(void)
{
  int N = 1000000;
  int steps = 50000;
  double delta_t = 0.001;
  double critical_distance = 0.5;
  double m1 = 10.;
  double m2 = 20.;
  double m3 = 30.;
  double *p1_x, *p1_y, *p1_z;
  double *p2_x, *p2_y, *p2_z;
  double *p3_x, *p3_y, *p3_z;
  double *p1_prime_x, *p1_prime_y, *p1_prime_z;
  double *p2_prime_x, *p2_prime_y, *p2_prime_z;
  double *p3_prime_x, *p3_prime_y, *p3_prime_z;
  double *dv_1_x, *dv_1_y, *dv_1_z;
  double *dv_2_x, *dv_2_y, *dv_2_z;
  double *dv_3_x, *dv_3_y, *dv_3_z;
  double *dv_1pr_x, *dv_1pr_y, *dv_1pr_z;
  double *dv_2pr_x, *dv_2pr_y, *dv_2pr_z;
  double *dv_3pr_x, *dv_3pr_y, *dv_3pr_z;
  double *v1_x, *v1_y, *v1_z;
  double *v2_x, *v2_y, *v2_z;
  double *v3_x, *v3_y, *v3_z;
  double *v1_prime_x, *v1_prime_y, *v1_prime_z;
  double *v2_prime_x, *v2_prime_y, *v2_prime_z;
  double *v3_prime_x, *v3_prime_y, *v3_prime_z;
  double *nv1_x, *nv1_y, *nv1_z;
  double *nv2_x, *nv2_y, *nv2_z;
  double *nv3_x, *nv3_y, *nv3_z;
  double *nv1_prime_x, *nv1_prime_y, *nv1_prime_z;
  double *nv2_prime_x, *nv2_prime_y, *nv2_prime_z;
  double *nv3_prime_x, *nv3_prime_y, *nv3_prime_z;

  double *d_p1_x, *d_p1_y, *d_p1_z;
  double *d_p2_x, *d_p2_y, *d_p2_z;
  double *d_p3_x, *d_p3_y, *d_p3_z;
  double *d_p1_prime_x, *d_p1_prime_y, *d_p1_prime_z;
  double *d_p2_prime_x, *d_p2_prime_y, *d_p2_prime_z;
  double *d_p3_prime_x, *d_p3_prime_y, *d_p3_prime_z;
  double *d_dv_1_x, *d_dv_1_y, *d_dv_1_z;
  double *d_dv_2_x, *d_dv_2_y, *d_dv_2_z;
  double *d_dv_3_x, *d_dv_3_y, *d_dv_3_z;
  double *d_dv_1pr_x, *d_dv_1pr_y, *d_dv_1pr_z;
  double *d_dv_2pr_x, *d_dv_2pr_y, *d_dv_2pr_z;
  double *d_dv_3pr_x, *d_dv_3pr_y, *d_dv_3pr_z;
  double *d_v1_x, *d_v1_y, *d_v1_z;
  double *d_v2_x, *d_v2_y, *d_v2_z;
  double *d_v3_x, *d_v3_y, *d_v3_z;
  double *d_v1_prime_x, *d_v1_prime_y, *d_v1_prime_z;
  double *d_v2_prime_x, *d_v2_prime_y, *d_v2_prime_z;
  double *d_v3_prime_x, *d_v3_prime_y, *d_v3_prime_z;
  double *d_nv1_x, *d_nv1_y, *d_nv1_z;
  double *d_nv2_x, *d_nv2_y, *d_nv2_z;
  double *d_nv3_x, *d_nv3_y, *d_nv3_z;
  double *d_nv1_prime_x, *d_nv1_prime_y, *d_nv1_prime_z;
  double *d_nv2_prime_x, *d_nv2_prime_y, *d_nv2_prime_z;
  double *d_nv3_prime_x, *d_nv3_prime_y, *d_nv3_prime_z;

  bool *still_together, *d_still_together;
  int *times, *d_times;
  bool *not_diverged, *d_not_diverged;

  p1_x = (double*)malloc(N*sizeof(double));
  p1_y = (double*)malloc(N*sizeof(double));
  p1_z = (double*)malloc(N*sizeof(double));

  p2_x = (double*)malloc(N*sizeof(double));
  p2_y = (double*)malloc(N*sizeof(double));
  p2_z = (double*)malloc(N*sizeof(double));

  p3_x = (double*)malloc(N*sizeof(double));
  p3_y = (double*)malloc(N*sizeof(double));
  p3_z = (double*)malloc(N*sizeof(double));

  p1_prime_x = (double*)malloc(N*sizeof(double));
  p1_prime_y = (double*)malloc(N*sizeof(double));
  p1_prime_z = (double*)malloc(N*sizeof(double));

  p2_prime_x = (double*)malloc(N*sizeof(double));
  p2_prime_y = (double*)malloc(N*sizeof(double));
  p2_prime_z = (double*)malloc(N*sizeof(double));

  p3_prime_x = (double*)malloc(N*sizeof(double));
  p3_prime_y = (double*)malloc(N*sizeof(double));
  p3_prime_z = (double*)malloc(N*sizeof(double));

  dv_1_x = (double*)malloc(N*sizeof(double));
  dv_1_y = (double*)malloc(N*sizeof(double));
  dv_1_z = (double*)malloc(N*sizeof(double));

  dv_2_x = (double*)malloc(N*sizeof(double));
  dv_2_y = (double*)malloc(N*sizeof(double));
  dv_2_z = (double*)malloc(N*sizeof(double));

  dv_3_x = (double*)malloc(N*sizeof(double));
  dv_3_y = (double*)malloc(N*sizeof(double));
  dv_3_z = (double*)malloc(N*sizeof(double));

  dv_1pr_x = (double*)malloc(N*sizeof(double));
  dv_1pr_y = (double*)malloc(N*sizeof(double));
  dv_1pr_z = (double*)malloc(N*sizeof(double));

  dv_2pr_x = (double*)malloc(N*sizeof(double));
  dv_2pr_y = (double*)malloc(N*sizeof(double));
  dv_2pr_z = (double*)malloc(N*sizeof(double));

  dv_3pr_x = (double*)malloc(N*sizeof(double));
  dv_3pr_y = (double*)malloc(N*sizeof(double));
  dv_3pr_z = (double*)malloc(N*sizeof(double));

  v1_x = (double*)malloc(N*sizeof(double));
  v1_y = (double*)malloc(N*sizeof(double));
  v1_z = (double*)malloc(N*sizeof(double));

  v2_x = (double*)malloc(N*sizeof(double));
  v2_y = (double*)malloc(N*sizeof(double));
  v2_z = (double*)malloc(N*sizeof(double));

  v3_x = (double*)malloc(N*sizeof(double));
  v3_y = (double*)malloc(N*sizeof(double));
  v3_z = (double*)malloc(N*sizeof(double));

  v1_prime_x = (double*)malloc(N*sizeof(double));  
  v1_prime_y = (double*)malloc(N*sizeof(double));
  v1_prime_z = (double*)malloc(N*sizeof(double));

  v2_prime_x = (double*)malloc(N*sizeof(double));  
  v2_prime_y = (double*)malloc(N*sizeof(double));
  v2_prime_z = (double*)malloc(N*sizeof(double));

  v3_prime_x = (double*)malloc(N*sizeof(double));  
  v3_prime_y = (double*)malloc(N*sizeof(double));
  v3_prime_z = (double*)malloc(N*sizeof(double));

  nv1_x = (double*)malloc(N*sizeof(double));
  nv1_y = (double*)malloc(N*sizeof(double));
  nv1_z = (double*)malloc(N*sizeof(double));

  nv2_x = (double*)malloc(N*sizeof(double));
  nv2_y = (double*)malloc(N*sizeof(double));
  nv2_z = (double*)malloc(N*sizeof(double));

  nv3_x = (double*)malloc(N*sizeof(double));
  nv3_y = (double*)malloc(N*sizeof(double));
  nv3_z = (double*)malloc(N*sizeof(double));

  nv1_prime_x = (double*)malloc(N*sizeof(double));
  nv1_prime_y = (double*)malloc(N*sizeof(double));
  nv1_prime_z = (double*)malloc(N*sizeof(double));

  nv2_prime_x = (double*)malloc(N*sizeof(double));
  nv2_prime_y = (double*)malloc(N*sizeof(double));
  nv2_prime_z = (double*)malloc(N*sizeof(double));
  
  nv3_prime_x = (double*)malloc(N*sizeof(double));
  nv3_prime_y = (double*)malloc(N*sizeof(double));
  nv3_prime_z = (double*)malloc(N*sizeof(double));

  still_together = (bool*)malloc(N*sizeof(bool));
  times = (int*)malloc(N*sizeof(int));
  not_diverged = (bool*)malloc(N*sizeof(bool));  

  hipMalloc(&d_p1_x, N*sizeof(double)); 
  hipMalloc(&d_p1_y, N*sizeof(double)); 
  hipMalloc(&d_p1_z, N*sizeof(double)); 

  hipMalloc(&d_p2_x, N*sizeof(double));
  hipMalloc(&d_p2_y, N*sizeof(double));
  hipMalloc(&d_p2_z, N*sizeof(double));

  hipMalloc(&d_p3_x, N*sizeof(double));
  hipMalloc(&d_p3_y, N*sizeof(double));
  hipMalloc(&d_p3_z, N*sizeof(double));

  hipMalloc(&d_p1_prime_x, N*sizeof(double));
  hipMalloc(&d_p1_prime_y, N*sizeof(double));
  hipMalloc(&d_p1_prime_z, N*sizeof(double));

  hipMalloc(&d_p2_prime_x, N*sizeof(double));
  hipMalloc(&d_p2_prime_y, N*sizeof(double));
  hipMalloc(&d_p2_prime_z, N*sizeof(double));

  hipMalloc(&d_p3_prime_x, N*sizeof(double));
  hipMalloc(&d_p3_prime_y, N*sizeof(double));
  hipMalloc(&d_p3_prime_z, N*sizeof(double));

  hipMalloc(&d_dv_1_x, N*sizeof(double));
  hipMalloc(&d_dv_1_y, N*sizeof(double));
  hipMalloc(&d_dv_1_z, N*sizeof(double));

  hipMalloc(&d_dv_2_x, N*sizeof(double));
  hipMalloc(&d_dv_2_y, N*sizeof(double));
  hipMalloc(&d_dv_2_z, N*sizeof(double));

  hipMalloc(&d_dv_3_x, N*sizeof(double));
  hipMalloc(&d_dv_3_y, N*sizeof(double));
  hipMalloc(&d_dv_3_z, N*sizeof(double));

  hipMalloc(&d_dv_1pr_x, N*sizeof(double));
  hipMalloc(&d_dv_1pr_y, N*sizeof(double));
  hipMalloc(&d_dv_1pr_z, N*sizeof(double));

  hipMalloc(&d_dv_2pr_x, N*sizeof(double));
  hipMalloc(&d_dv_2pr_y, N*sizeof(double));
  hipMalloc(&d_dv_2pr_z, N*sizeof(double));

  hipMalloc(&d_dv_3pr_x, N*sizeof(double));
  hipMalloc(&d_dv_3pr_y, N*sizeof(double));
  hipMalloc(&d_dv_3pr_z, N*sizeof(double));

  hipMalloc(&d_v1_x, N*sizeof(double));
  hipMalloc(&d_v1_y, N*sizeof(double));
  hipMalloc(&d_v1_z, N*sizeof(double));

  hipMalloc(&d_v2_x, N*sizeof(double));
  hipMalloc(&d_v2_y, N*sizeof(double));
  hipMalloc(&d_v2_z, N*sizeof(double));

  hipMalloc(&d_v3_x, N*sizeof(double));
  hipMalloc(&d_v3_y, N*sizeof(double));
  hipMalloc(&d_v3_z, N*sizeof(double));

  hipMalloc(&d_v1_prime_x, N*sizeof(double));
  hipMalloc(&d_v1_prime_y, N*sizeof(double));
  hipMalloc(&d_v1_prime_z, N*sizeof(double));

  hipMalloc(&d_v2_prime_x, N*sizeof(double));
  hipMalloc(&d_v2_prime_y, N*sizeof(double));
  hipMalloc(&d_v2_prime_z, N*sizeof(double));

  hipMalloc(&d_v3_prime_x, N*sizeof(double));
  hipMalloc(&d_v3_prime_y, N*sizeof(double));
  hipMalloc(&d_v3_prime_z, N*sizeof(double));

  hipMalloc(&d_nv1_x, N*sizeof(double));
  hipMalloc(&d_nv1_y, N*sizeof(double));
  hipMalloc(&d_nv1_z, N*sizeof(double));

  hipMalloc(&d_nv2_x, N*sizeof(double));
  hipMalloc(&d_nv2_y, N*sizeof(double));
  hipMalloc(&d_nv2_z, N*sizeof(double));

  hipMalloc(&d_nv3_x, N*sizeof(double));
  hipMalloc(&d_nv3_y, N*sizeof(double));
  hipMalloc(&d_nv3_z, N*sizeof(double));

  hipMalloc(&d_nv1_prime_x, N*sizeof(double));
  hipMalloc(&d_nv1_prime_y, N*sizeof(double));
  hipMalloc(&d_nv1_prime_z, N*sizeof(double));

  hipMalloc(&d_nv2_prime_x, N*sizeof(double));
  hipMalloc(&d_nv2_prime_y, N*sizeof(double));
  hipMalloc(&d_nv2_prime_z, N*sizeof(double));

  hipMalloc(&d_nv3_prime_x, N*sizeof(double));
  hipMalloc(&d_nv3_prime_y, N*sizeof(double));
  hipMalloc(&d_nv3_prime_z, N*sizeof(double));

  hipMalloc(&d_still_together, N*sizeof(bool));
  hipMalloc(&d_times, N*sizeof(int));
  hipMalloc(&d_not_diverged, N*sizeof(bool));

  int resolution = sqrt(N);
  double range = 40;
  double step_size = range / resolution;
  for (int i = 0; i < N; i++) {
    int remainder = i % resolution;
    int step = i / resolution;
    p1_x[i] = -20. + 40*(double(remainder)/double(resolution));
    p1_y[i] = -20. + 40*(double(step)/double(resolution));
    p1_z[i] = -11.0;

    p2_x[i] = 0.0;
    p2_y[i] = 0.0;
    p2_z[i] = 0.0;

    p3_x[i] = 10.0;
    p3_y[i] = 10.0;
    p3_z[i] = 12.0;

    // shift p1 in all x, y, z vectors
    p1_prime_x[i] = -20. + 40*(double(remainder)/double(resolution)) + 0.001;
    p1_prime_y[i] = -20. + 40*(double(step)/double(resolution)) + 0.001;
    p1_prime_z[i] = -11.0 + 0.001;

    p2_prime_x[i] = 0.0;
    p2_prime_y[i] = 0.0;
    p2_prime_z[i] = 0.0;

    p3_prime_x[i] = 10.0;
    p3_prime_y[i] = 10.0;
    p3_prime_z[i] = 12.0;

    v1_x[i] = -3.;
    v1_y[i] = 0.;
    v1_z[i] = 0.;

    v2_x[i] = 0.;
    v2_y[i] = 0.;
    v2_z[i] = 0.;

    v3_x[i] = 3.;
    v3_y[i] = 0.;
    v3_z[i] = 0.;

    v1_prime_x[i] = -3.;
    v1_prime_y[i] = 0.;
    v1_prime_z[i] = 0.;

    v2_prime_x[i] = 0.;
    v2_prime_y[i] = 0.;
    v2_prime_z[i] = 0.;

    v3_prime_x[i] = 3.;
    v3_prime_y[i] = 0.;
    v3_prime_z[i] = 0.;

    times[i] = 0;
    still_together[i] = true;
    not_diverged[i] = true;
  }

  hipMemcpy(d_p1_x, p1_x, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_p1_y, p1_y, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_p1_z, p1_z, N*sizeof(double), hipMemcpyHostToDevice);

  hipMemcpy(d_p2_x, p2_x, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_p2_y, p2_y, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_p2_z, p2_z, N*sizeof(double), hipMemcpyHostToDevice);

  hipMemcpy(d_p3_x, p3_x, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_p3_y, p3_y, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_p3_z, p3_z, N*sizeof(double), hipMemcpyHostToDevice);

  hipMemcpy(d_p1_prime_x, p1_prime_x, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_p1_prime_y, p1_prime_y, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_p1_prime_z, p1_prime_z, N*sizeof(double), hipMemcpyHostToDevice);

  hipMemcpy(d_p2_prime_x, p2_prime_x, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_p2_prime_y, p2_prime_y, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_p2_prime_z, p2_prime_z, N*sizeof(double), hipMemcpyHostToDevice);

  hipMemcpy(d_p3_prime_x, p3_prime_x, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_p3_prime_y, p3_prime_y, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_p3_prime_z, p3_prime_z, N*sizeof(double), hipMemcpyHostToDevice);

  hipMemcpy(d_dv_1_x, dv_1_x, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_dv_1_y, dv_1_y, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_dv_1_z, dv_1_z, N*sizeof(double), hipMemcpyHostToDevice);

  hipMemcpy(d_dv_2_x, dv_2_x, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_dv_2_y, dv_2_y, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_dv_2_z, dv_2_z, N*sizeof(double), hipMemcpyHostToDevice);

  hipMemcpy(d_dv_3_x, dv_3_x, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_dv_3_y, dv_3_y, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_dv_3_z, dv_3_z, N*sizeof(double), hipMemcpyHostToDevice);

  hipMemcpy(d_dv_1pr_x, dv_1pr_x, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_dv_1pr_y, dv_1pr_y, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_dv_1pr_z, dv_1pr_z, N*sizeof(double), hipMemcpyHostToDevice);

  hipMemcpy(d_dv_2pr_x, dv_2pr_x, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_dv_2pr_y, dv_2pr_y, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_dv_2pr_z, dv_2pr_z, N*sizeof(double), hipMemcpyHostToDevice);

  hipMemcpy(d_dv_3pr_x, dv_3pr_x, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_dv_3pr_y, dv_3pr_y, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_dv_3pr_z, dv_3pr_z, N*sizeof(double), hipMemcpyHostToDevice);

  hipMemcpy(d_v1_x, v1_x, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_v1_y, v1_y, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_v1_z, v1_z, N*sizeof(double), hipMemcpyHostToDevice);

  hipMemcpy(d_v2_x, v2_x, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_v2_y, v2_y, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_v2_z, v2_z, N*sizeof(double), hipMemcpyHostToDevice);

  hipMemcpy(d_v3_x, v3_x, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_v3_y, v3_y, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_v3_z, v3_z, N*sizeof(double), hipMemcpyHostToDevice);

  hipMemcpy(d_v1_prime_x, v1_prime_x, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_v1_prime_y, v1_prime_y, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_v1_prime_z, v1_prime_z, N*sizeof(double), hipMemcpyHostToDevice);

  hipMemcpy(d_v2_prime_x, v2_prime_x, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_v2_prime_y, v2_prime_y, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_v2_prime_z, v2_prime_z, N*sizeof(double), hipMemcpyHostToDevice);

  hipMemcpy(d_v3_prime_x, v3_prime_x, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_v3_prime_y, v3_prime_y, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_v3_prime_z, v3_prime_z, N*sizeof(double), hipMemcpyHostToDevice);

  hipMemcpy(d_nv1_x, nv1_x, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_nv1_y, nv1_y, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_nv1_z, nv1_z, N*sizeof(double), hipMemcpyHostToDevice);

  hipMemcpy(d_nv2_x, nv2_x, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_nv2_y, nv2_y, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_nv2_z, nv2_z, N*sizeof(double), hipMemcpyHostToDevice);

  hipMemcpy(d_nv3_x, nv3_x, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_nv3_y, nv3_y, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_nv3_z, nv3_z, N*sizeof(double), hipMemcpyHostToDevice);

  hipMemcpy(d_nv1_prime_x, nv1_prime_x, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_nv1_prime_y, nv1_prime_y, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_nv1_prime_z, nv1_prime_z, N*sizeof(double), hipMemcpyHostToDevice);

  hipMemcpy(d_nv2_prime_x, nv2_prime_x, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_nv2_prime_y, nv2_prime_y, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_nv2_prime_z, nv2_prime_z, N*sizeof(double), hipMemcpyHostToDevice);

  hipMemcpy(d_nv3_prime_x, nv3_prime_x, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_nv3_prime_y, nv3_prime_y, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_nv3_prime_z, nv3_prime_z, N*sizeof(double), hipMemcpyHostToDevice);

  hipMemcpy(d_times, times, N*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_still_together, still_together, N*sizeof(bool), hipMemcpyHostToDevice);
  hipMemcpy(d_not_diverged, not_diverged, N*sizeof(bool), hipMemcpyHostToDevice);

  // initialize timer
  std::chrono::time_point<std::chrono::system_clock> start, end;
  start = std::chrono::system_clock::now();

  // call CUDA kernal on inputs in configuration <<< blockIdx, threadIdx>>>>
  divergence<<<(N+127)/128, 128>>>(
      N, 
      steps, 
      delta_t,
      d_still_together,
      d_not_diverged,
      d_times,
      m1, m2, m3,
      critical_distance,
      d_p1_x, d_p1_y, d_p1_z, 
      d_p2_x, d_p2_y, d_p2_z, 
      d_p3_x, d_p3_y, d_p3_z, 
      d_p1_prime_x, d_p1_prime_y, d_p1_prime_z, 
      d_p2_prime_x, d_p2_prime_y, d_p2_prime_z, 
      d_p3_prime_x, d_p3_prime_y, d_p3_prime_z,
      d_dv_1_x, d_dv_1_y, d_dv_1_z,
      d_dv_2_x, d_dv_2_y, d_dv_2_z,
      d_dv_3_x, d_dv_3_y, d_dv_3_z,
      d_dv_1pr_x, d_dv_1pr_y, d_dv_1pr_z,
      d_dv_2pr_x, d_dv_2pr_y, d_dv_2pr_z,
      d_dv_3pr_x, d_dv_3pr_y, d_dv_3pr_z,
      d_v1_x, d_v1_y, d_v1_z,
      d_v2_x, d_v2_y, d_v2_z,
      d_v3_x, d_v3_y, d_v3_z,
      d_v1_prime_x, d_v1_prime_y, d_v1_prime_z,
      d_v2_prime_x, d_v2_prime_y, d_v2_prime_z,
      d_v3_prime_x, d_v3_prime_y, d_v3_prime_z,
      d_nv1_x, d_nv1_y, d_nv1_z,
      d_nv2_x, d_nv2_y, d_nv2_z,
      d_nv3_x, d_nv3_y, d_nv3_z,
      d_nv1_prime_x, d_nv1_prime_y, d_nv1_prime_z,    
      d_nv2_prime_x, d_nv2_prime_y, d_nv2_prime_z,
      d_nv3_prime_x, d_nv3_prime_y, d_nv3_prime_z
      );

  hipDeviceSynchronize();

  // measure elapsed kernal runtime
  end = std::chrono::system_clock::now();
  std::chrono::duration<double> elapsed_seconds = end - start;
  std::time_t end_time = std::chrono::system_clock::to_time_t(end);
  std::cout << "Elapsed Time: " << elapsed_seconds.count() << "s\n";

  hipMemcpy(times, d_times, N*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(still_together, d_still_together, N*sizeof(bool), hipMemcpyDeviceToHost);
  hipMemcpy(not_diverged, d_not_diverged, N*sizeof(bool), hipMemcpyDeviceToHost);
  hipMemcpy(p1_x, d_p1_x, N*sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(p1_y, d_p1_y, N*sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(p1_z, d_p1_z, N*sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(p1_prime_x, d_p1_prime_x, N*sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(p1_prime_y, d_p1_prime_y, N*sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(p1_prime_z, d_p1_prime_z, N*sizeof(double), hipMemcpyDeviceToHost);

  // check computation for completion and accuracy
  for (int k=0; k<2; k++) {
    std::cout << times[k] << ' ';
    std::cout << still_together[k] << ' ';
    std::cout << not_diverged[k] << '\n';
    std::cout << p1_x[k] << ' ';
    std::cout << p1_y[k] << ' ';
    std::cout << p1_z[k] << ' ';
    std::cout << p1_prime_x[k] << ' ';
    std::cout << p1_prime_y[k] << ' ';
    std::cout << p1_prime_z[k] << ' ';
    std::cout << '\n';
  }

  hipFree(d_p1_x); hipFree(d_p1_y); hipFree(d_p1_z);
  hipFree(d_p2_x); hipFree(d_p2_y); hipFree(d_p2_z);
  hipFree(d_p3_x); hipFree(d_p3_y); hipFree(d_p3_z);

  hipFree(d_p1_prime_x); hipFree(d_p1_prime_y); hipFree(d_p1_prime_z);  
  hipFree(d_p2_prime_x); hipFree(d_p2_prime_y); hipFree(d_p2_prime_z);
  hipFree(d_p3_prime_x); hipFree(d_p3_prime_y); hipFree(d_p3_prime_z);

  hipFree(d_dv_1_x); hipFree(d_dv_1_y); hipFree(d_dv_1_z);
  hipFree(d_dv_2_x); hipFree(d_dv_2_y); hipFree(d_dv_2_z);
  hipFree(d_dv_3_x); hipFree(d_dv_3_y); hipFree(d_dv_3_z);

  hipFree(d_dv_1pr_x); hipFree(d_dv_1pr_y); hipFree(d_dv_1pr_z);
  hipFree(d_dv_2pr_x); hipFree(d_dv_2pr_y); hipFree(d_dv_2pr_z);
  hipFree(d_dv_3pr_x); hipFree(d_dv_3pr_y); hipFree(d_dv_3pr_z);

  hipFree(d_nv1_x); hipFree(d_nv1_y); hipFree(d_nv1_z);
  hipFree(d_nv2_x); hipFree(d_nv2_y); hipFree(d_nv2_z);
  hipFree(d_nv3_x); hipFree(d_nv3_y); hipFree(d_nv3_z);

  hipFree(d_v1_x); hipFree(d_v1_y); hipFree(d_v1_z);
  hipFree(d_v2_x); hipFree(d_v2_y); hipFree(d_v2_z);
  hipFree(d_v3_x); hipFree(d_v3_y); hipFree(d_v3_z);

  hipFree(d_v1_prime_x); hipFree(d_v1_prime_y); hipFree(d_v1_prime_z);
  hipFree(d_v2_prime_x); hipFree(d_v2_prime_y); hipFree(d_v2_prime_z);
  hipFree(d_v3_prime_x); hipFree(d_v3_prime_y); hipFree(d_v3_prime_z);

  hipFree(d_nv1_prime_x); hipFree(d_nv1_prime_y); hipFree(d_nv1_prime_z);
  hipFree(d_nv2_prime_x); hipFree(d_nv2_prime_y); hipFree(d_nv2_prime_z);
  hipFree(d_nv3_prime_x); hipFree(d_nv3_prime_y); hipFree(d_nv3_prime_z);

  hipFree(d_times); hipFree(d_still_together); hipFree(d_not_diverged);

  free(p1_x); free(p1_y); free(p1_z);
  free(p2_x); free(p2_y); free(p2_z);
  free(p3_x); free(p3_y); free(p3_z);

  free(p1_prime_x); free(p1_prime_y); free(p1_prime_z);  
  free(p2_prime_x); free(p2_prime_y); free(p2_prime_z);  
  free(p3_prime_x); free(p3_prime_y); free(p3_prime_z);  

  free(dv_1_x); free(dv_1_y); free(dv_1_z);
  free(dv_2_x); free(dv_2_y); free(dv_2_z);
  free(dv_3_x); free(dv_3_y); free(dv_3_z);

  free(dv_1pr_x); free(dv_1pr_y); free(dv_1pr_z);
  free(dv_2pr_x); free(dv_2pr_y); free(dv_2pr_z);
  free(dv_3pr_x); free(dv_3pr_y); free(dv_3pr_z);

  free(v1_x); free(v1_y); free(v1_z);
  free(v2_x); free(v2_y); free(v2_z);
  free(v3_x); free(v3_y); free(v3_z);

  free(v1_prime_x); free(v1_prime_y); free(v1_prime_z);
  free(v2_prime_x); free(v2_prime_y); free(v2_prime_z);
  free(v3_prime_x); free(v3_prime_y); free(v3_prime_z);

  free(nv1_x); free(nv1_y); free(nv1_z);
  free(nv2_x); free(nv2_y); free(nv2_z);
  free(nv3_x); free(nv3_y); free(nv3_z);

  free(nv1_prime_x); free(nv1_prime_y); free(nv1_prime_z);
  free(nv2_prime_x); free(nv2_prime_y); free(nv2_prime_z);
  free(nv3_prime_x); free(nv3_prime_y); free(nv3_prime_z);


  free(times); free(still_together); free(not_diverged);

}















