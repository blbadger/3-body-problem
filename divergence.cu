
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

__global__
void saxpy(int n, float *p1, float *p2, float *p3, float *p1_prime, float *p2_prime, float *p3_prime, int *diverged)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  for (int j=0; j < 50000; j++) {
    if (i < n) {
      if (p1[i] - p1_prime[i] > 1 || p2[i] - p2_prime[i] > 1 || p3[i] - p3_prime[i] > 1){
        diverged[i] = i;
      } 
    }

  }
}

int main(void)
{
  int N = 1<<18;
  std::cout << N;
  float *p1, *p2, *p3, *p1_prime, *p2_prime, *p3_prime, *d_p1, *d_p2, *d_p3, *d_p1_prime, *d_p2_prime, *d_p3_prime;
  int *diverged, *d_diverged;
  p1 = (float*)malloc(N*sizeof(float));
  p2 = (float*)malloc(N*sizeof(float));
  p3 = (float*)malloc(N*sizeof(float));

  p1_prime = (float*)malloc(N*sizeof(float));
  p2_prime = (float*)malloc(N*sizeof(float));
  p3_prime = (float*)malloc(N*sizeof(float));
  diverged = (int*)malloc(N*sizeof(int));

  hipMalloc(&d_p1, N*sizeof(float)); 
  hipMalloc(&d_p2, N*sizeof(float));
  hipMalloc(&d_p3, N*sizeof(float));
  hipMalloc(&d_p1_prime, N*sizeof(float));
  hipMalloc(&d_p2_prime, N*sizeof(float));
  hipMalloc(&d_p3_prime, N*sizeof(float));
  hipMalloc(&d_diverged, N*sizeof(int));

  for (int i = 0; i < N; i++) {
    p1[i] = 0.0f + i;
    p2[i] = 0.0f + i;
    p3[i] = 0.0f + i;
    p1_prime[i] = 0.0f + i + 0.0001f;
    p2_prime[i] = 0.0f + i;
    p3_prime[i] = 0.0f + i;
    diverged[i] = 0;
  }

  hipMemcpy(d_p1, p1, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_p2, p2, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_p3, p3, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_p1_prime, p1_prime, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_p2_prime, p2_prime, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_p3_prime, p3_prime, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_diverged, diverged, N*sizeof(int), hipMemcpyHostToDevice);

  // call CUDA kernal
  saxpy<<<(N+255)/256, 256>>>(N, d_p1, d_p2, d_p3, d_p1_prime, d_p2_prime, d_p3_prime, d_diverged);

  hipMemcpy(diverged, d_diverged, N*sizeof(float), hipMemcpyDeviceToHost);
  for (int i=0; i < 100; i++) {
    std::cout<< diverged[i];
    std::cout << ' ';
  }

  hipFree(d_p1);
  hipFree(d_p3);
  hipFree(d_p3);
  hipFree(d_p1_prime);
  hipFree(d_p2_prime);
  hipFree(d_p3_prime);

  free(p1);
  free(p2);
  free(p3);
}